#include "hip/hip_runtime.h"
//
// Created by ameen on 07/05/20.
//

#include "sql_delete.cuh"

#define NUM_THREADS 512

__global__ void
deleteKernel(void *data, int rowSize, int *offset, int offsetSize, ColType *types, myExpr *exprs, int numRows, bool *flag_d) {
    void *res;
    int resType = 1;
    int rowsPerBlock = (numRows + NUM_THREADS - 1) / NUM_THREADS;
    unsigned int start = rowsPerBlock * threadIdx.x;
    unsigned int end = rowsPerBlock * (threadIdx.x + 1);

    void *tempRow = malloc(rowSize);

    void *row;
    bool flag;
    for (unsigned int i = start; i < end; ++i) {
        if (i >= numRows) break;
        row = (char *)data + i * rowSize;
        eval(row, offset, types, exprs, res, resType);
        flag = false;
        if (resType == RESTYPE_INT) {
            flag = *(int *) res != 0;
        } else if (resType == RESTYPE_FLT) {
            flag = *(float *) res != 0;
        }
        free(res);
        if (!flag) continue;
        flag_d[i] = flag;
    }
}

void sql_delete::execute(std::string &query) {
    hsql::SQLParserResult *result = hsql::SQLParser::parseSQLString(query);
    std::vector<std::string> columnNames;
    std::string tableName;

    if (result->isValid()) {
        const auto *stmt = (const hsql::DeleteStatement *) result->getStatement(0);
        tableName = stmt->tableName;
        std::vector<myExpr> flattenedExpr;
        Data d(tableName);
        exprToVec(stmt->expr, flattenedExpr, d.mdata.columns, d);

        hipSetDevice(0);
        hipDeviceReset();

        int rowSize = d.mdata.rowSize;
        void *data = malloc(d.chunkSize * rowSize);
        void *data_d;
        int numCols = d.mdata.columns.size();
        ColType *type_d;
        hipMalloc(&type_d, sizeof(ColType) * numCols);
        hipMemcpy(type_d, &d.mdata.datatypes[0], sizeof(ColType) * numCols, hipMemcpyHostToDevice);
        myExpr *where_d;
        hipMalloc(&where_d, sizeof(myExpr) * flattenedExpr.size());
        hipMemcpy(where_d, &flattenedExpr[0], sizeof(myExpr) * flattenedExpr.size(), hipMemcpyHostToDevice);
        int *offsets = (int *) malloc(sizeof(int) * (numCols + 1));
        offsets[0] = 0; //d.mdata.datatypes[0].size;
        for (int i = 1; i <= numCols; i++) {
            offsets[i] = offsets[i - 1] + d.mdata.datatypes[i - 1].size;
        }
        int *offsets_d;
        hipMalloc(&offsets_d, sizeof(int) * (numCols + 1));
        hipMemcpy(offsets_d, offsets, sizeof(int) * (numCols + 1), hipMemcpyHostToDevice);
        int numRows = d.read(data);
        hipMalloc(&data_d, d.chunkSize * rowSize);
        bool *flag = (bool *)malloc(numRows * sizeof(bool));
        bool *flag_d;
        hipMalloc(&flag_d,numRows * sizeof(bool));
        d.mdata.rowCount = 0;
        while (numRows > 0) {
            hipMemcpy(data_d, data, rowSize * numRows, hipMemcpyHostToDevice);
            deleteKernel<<<1, NUM_THREADS>>>(data_d, rowSize, offsets_d, numCols, type_d, where_d, numRows, flag_d);
            hipDeviceSynchronize();
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                printf("Error at %d: %s\n", __LINE__, hipGetErrorString(err));
            }
//            hipMemcpy(data, data_d, rowSize * numRows, hipMemcpyDeviceToHost);
//            d.write(data, numRows * d.mdata.rowSize);

            hipMemcpy(flag, flag_d, numRows * sizeof(bool), hipMemcpyDeviceToHost);
            for (int k=0;k<numRows;k++)
                if(flag[k])
                    d.writeRow((char *)data+k*rowSize);
            numRows = d.read(data);
        }
        d.mdata.commit();
        //write to file after checking flag
        // Free all the data
        free(data);
        free(offsets);
        free(flag);
        hipFree(data_d);
        hipFree(type_d);
        hipFree(where_d);
        hipFree(offsets_d);
        hipFree(flag_d);
    } else {
        printf("QUERY is invalid\n");
    }
}
