#include "hip/hip_runtime.h"
//
// Created by gautam on 25/04/20.
//

#include "sql_select.cuh"

#define NUM_THREADS 512

__global__ void selectKernel(void *data, int rowSize, int *offset, int offsetSize, ColType *types, myExpr *exprs, int numRows) {
    int rowsPerBlock = (numRows + NUM_THREADS - 1) / NUM_THREADS;
    unsigned int start = rowsPerBlock * threadIdx.x;
    unsigned int end = rowsPerBlock * (threadIdx.x + 1);

    void *res;
    int resType = 1;
    void *row;
    bool flag;
    for (unsigned int i = start; i < end; i++) {
        row = (char *)data + i * rowSize;
        eval(row, offset, types, exprs, res, resType);
        if (i < numRows) {
            flag = false;
            if (resType == RESTYPE_INT) {
                flag = *(int *) res != 0;
            } else if (resType == RESTYPE_FLT) {
                flag = *(float *) res != 0;
            }
            free(res);
            if (!flag) continue;
            // Condition is satisfied, write code here
            printRowDevice(row, types, offsetSize);
        }
    }
}

__global__ void
joinKernel(void *left, void *right, void *join, myExpr *joinExpr, int *offset, int numCols, ColType *types, int rowSizeL, int rowSizeR, int numRowsL, int numRowsR,
           unsigned int *numRowsRes) {
    if (threadIdx.x == 0) {
        *numRowsRes = 0;
    }
    __syncthreads();
    int rowsPerThread = (numRowsL * numRowsR + NUM_THREADS - 1) / NUM_THREADS;
    const unsigned start = rowsPerThread * threadIdx.x;
    const unsigned end = rowsPerThread * (threadIdx.x + 1);
    const int rowSizeRes = rowSizeL + rowSizeR;

    void *res;
    int resType = 0;
    void *row;
    bool flag;
    unsigned old;

    for (unsigned i = start; i < end; ++i) {
        // row i in join is obtained from i / numRowsR from left and i % numRowsR in right
        unsigned l = i / numRowsR, r = i % numRowsR;
        if (l >= numRowsL || r >= numRowsR) break;
        row = malloc(rowSizeRes);
        memcpy(row, left, rowSizeL);
        memcpy((char *) row + rowSizeL, right, rowSizeR);
        // printRowDevice(row, types, offsetSize);
        // Error in eval
        eval(row, offset, types, joinExpr, res, resType);
        flag = false;
        if (resType == RESTYPE_INT) {
            flag = *(int *)res != 0;
        } else if (resType == RESTYPE_FLT) {
            flag = *(float *)res != 0;
        }
        free(res);
        if (!flag) continue;
        // Add this row to new table
        printRowDevice(row, types, numCols);
        old = atomicInc(numRowsRes, numRowsL * numRowsR);
        memcpy((char *) join + old * rowSizeRes, row, rowSizeRes);
    }
}


void sql_select::execute(std::string &query) {

    hsql::SQLParserResult *result;
    std::vector<std::string> columnNames;

    result = hsql::SQLParser::parseSQLString(query);
    columnNames = std::vector<std::string>();

    if(result->isValid()){
        const auto *stmt = (const hsql::SelectStatement *) result->getStatement(0);
        // Get column names
        for (hsql::Expr* expr : *stmt->selectList){
            switch (expr->type) {
                case hsql::kExprStar:
                    columnNames.emplace_back("*");
                    break;
                case hsql::kExprColumnRef:
                    columnNames.emplace_back(expr->name);
                    break;
                // case hsql::kExprTableColumnRef:
                // inprint(expr->table, expr->name, numIndent);
                // break;
                case hsql::kExprLiteralFloat:
                    columnNames.push_back(std::to_string(expr->fval));
                    break;
                case hsql::kExprLiteralInt:
                    columnNames.push_back(std::to_string(expr->ival));
                    break;
                case hsql::kExprLiteralString:
                    columnNames.emplace_back(expr->name);
                    break;
                // TODO: kExprFunctionRef (Distinct ?), kExprOperator (col1 + col2 ?)
                // case hsql::kExprFunctionRef:
                //     inprint(expr->name, numIndent);
                //     inprint(expr->expr->name, numIndent + 1);
                //     break;
                // case hsql::kExprOperator:
                //     printOperatorExpression(expr, numIndent);
                //     break;
                default:
                    fprintf(stderr, "Unrecognized expression type %d\n", expr->type);
                    return;
            }
        }
        // Get tables reference
        auto table = stmt->fromTable;
        Data *d;
        switch (table->type) {
            case hsql::kTableName:
                // inprint(table->name, numIndent);
                d = new Data(table->name);
                break;
            // case hsql::kTableSelect:
            //     // printSelectStatementInfo(table->select, numIndent);
            //     break;
            case hsql::kTableJoin: {
                //     // inprint("Join Table", numIndent);
                //     // inprint("Left", numIndent + 1);
                //     // printTableRefInfo(table->join->left, numIndent + 2);
                //     // inprint("Right", numIndent + 1);
                //     // printTableRefInfo(table->join->right, numIndent + 2);
                //     // inprint("Join Condition", numIndent + 1);
                //     // printExpression(table->join->condition, numIndent + 2);
                d = new Data(table->join->left->name, table->join->right->name);

                std::vector<myExpr> joinCondition;
                exprToVec(table->join->condition, joinCondition, d->mdata.columns);
                myExpr *joinCondition_d;
                hipMalloc(&joinCondition_d, joinCondition.size() * sizeof(myExpr));
                hipMemcpy(joinCondition_d, &joinCondition[0], sizeof(myExpr) * joinCondition.size(),
                           hipMemcpyHostToDevice);

                std::vector<int> offsets(d->mdata.columns.size() + 1);
                offsets[0] = 0;
                for (int i = 1; i <= d->mdata.columns.size(); ++i) {
                    offsets[i] = offsets[i - 1] + d->mdata.datatypes[i - 1].size;
                }
                int *offsets_d;
                hipMalloc(&offsets_d, sizeof(int) * ( d->mdata.columns.size() + 1));
                hipMemcpy(offsets_d, &offsets[0], sizeof(int) * (d->mdata.columns.size() + 1), hipMemcpyHostToDevice);

                ColType *type_d;
                hipMalloc(&type_d, sizeof(ColType) *  d->mdata.columns.size());
                hipMemcpy(type_d, &d->mdata.datatypes[0], sizeof(ColType) *  d->mdata.columns.size(), hipMemcpyHostToDevice);

                Data dL(table->join->left->name);
                dL.chunkSize = d->chunkSize;
                Data dR(table->join->right->name);
                dR.chunkSize = d->chunkSize;
                void *join = malloc(d->chunkSize * d->chunkSize * d->mdata.rowSize), *join_d; // Upto n^2 rows can be stored
                void *dataL = malloc(d->chunkSize * dL.mdata.rowSize), *dataL_d;
                void *dataR = malloc(d->chunkSize * dR.mdata.rowSize), *dataR_d;
                hipMalloc(&join_d, d->chunkSize * d->chunkSize * d->mdata.rowSize);
                hipMalloc(&dataL_d, dL.chunkSize * dL.mdata.rowSize);
                hipMalloc(&dataR_d, dR.chunkSize * dR.mdata.rowSize);
                int rowsReadL = dL.read(dataL), rowsReadR;
                hipMemcpy(dataL_d, dataL, rowsReadL * dL.mdata.rowSize, hipMemcpyHostToDevice);
                unsigned int numRowsJoin = 0;
                unsigned int *numRowsJoin_d;
                hipMalloc(&numRowsJoin_d, sizeof(unsigned int));

                // for (auto leaf : joinCondition) {
                //     printf("TYPE: %d, ival: %ld, fval: %f, sval: %s, left: %d, right: %d\n", leaf.type, leaf.iVal, leaf.fVal,
                //            leaf.sVal, leaf.childLeft, leaf.childRight);
                // }
                // for (int i = 0; i <= d->mdata.columns.size(); ++i) {
                //     printf("%d ", offsets[i]);
                // }
                // printf("%zu\n", d->mdata.columns.size());
                // printf("\n");
                // for (int i = 0; i < d->mdata.columns.size(); ++i) {
                //     printf("(%d, %d)", d->mdata.datatypes[i].type, d->mdata.datatypes[i].size);
                // }
                // printf("\n");
                // printf("%d\n", dL.mdata.rowSize);
                // printf("%d\n", dR.mdata.rowSize);

                while (rowsReadL > 0) {
                    // TODO: implement resetRead()
                    dR.restartRead();
                    rowsReadR = dR.read(dataR);
                    hipMemcpy(dataR_d, dataR, rowsReadR * dR.mdata.rowSize, hipMemcpyHostToDevice);
                    while (rowsReadR > 0) {
                        joinKernel<<<1, 512>>>(dataL_d, dataR_d, join_d, joinCondition_d, offsets_d, d->mdata.columns.size(),
                                               type_d, dL.mdata.rowSize, dR.mdata.rowSize, rowsReadL, rowsReadR,
                                               numRowsJoin_d);
                        hipDeviceSynchronize();
                        hipError_t err = hipGetLastError();
                        if (err != hipSuccess) {
                            printf("Error at %d: %s\n", __LINE__, hipGetErrorString(err));
                        }
                        hipMemcpy(join, join_d, numRowsJoin, hipMemcpyDeviceToHost);
                        hipMemcpy(&numRowsJoin, numRowsJoin_d, sizeof(unsigned int), hipMemcpyDeviceToHost);
                        d->write(join, (int)numRowsJoin * d->mdata.rowSize);
                        rowsReadR = dR.read(dataR);
                        hipMemcpy(dataR_d, dataR, rowsReadR * dR.mdata.rowSize, hipMemcpyHostToDevice);
                    }
                    rowsReadL = dL.read(dataL);
                    hipMemcpy(dataL_d, dataL, rowsReadL * dL.mdata.rowSize, hipMemcpyHostToDevice);
                }
                printf("_________________________________\n");
                break;
            }
            // case hsql::kTableCrossProduct:
            //     // for (TableRef* tbl : *table->list) printTableRefInfo(tbl, numIndent);
            //     break;
            default:
                printf("Will be handled later\n");
                return;
        }
        if (stmt->whereClause != nullptr) {
            // Get where
            printf("Where clause");
            std::vector<myExpr> tree;

            auto expr = stmt->whereClause;
            exprToVec(expr, tree, d->mdata.columns);
            free(expr);

            int rowSize = d->mdata.rowSize;
            void *data = malloc(d->chunkSize * rowSize);
            void *data_d;
            int numCols = d->mdata.columns.size();
            hipSetDevice(0);
            hipDeviceReset();

            ColType *type_d;
            hipMalloc(&type_d, sizeof(ColType) * numCols);
            hipMemcpy(type_d, &d->mdata.datatypes[0], sizeof(ColType) * numCols, hipMemcpyHostToDevice);
            myExpr *where_d;
            hipMalloc(&where_d, sizeof(myExpr) * tree.size());
            hipMemcpy(where_d, &tree[0], sizeof(myExpr) * tree.size(), hipMemcpyHostToDevice);
            int *offsets = (int *) malloc(sizeof(int) * (numCols + 1));
            offsets[0] = 0;
            for (int i = 1; i <= numCols; i++) {
                offsets[i] = offsets[i - 1] + d->mdata.datatypes[i - 1].size;
            }
            int *offsets_d;
            hipMalloc(&offsets_d, sizeof(int) * (numCols + 1));
            hipMemcpy(offsets_d, offsets, sizeof(int) * (numCols + 1), hipMemcpyHostToDevice);
            int numRows = d->read(data);

            // printing data in table
            // utils::printMultiple(data, d.mdata.datatypes, d.mdata.rowSize, d.mdata.rowCount);

            hipMalloc(&data_d, d->chunkSize * rowSize);
            while (numRows > 0) {
                // printf("Inside\n");
                // fflush(stdout);
                hipMemcpy(data_d, data, rowSize * numRows, hipMemcpyHostToDevice);
                selectKernel<<<1, NUM_THREADS>>>(data_d, rowSize, offsets_d, numCols, type_d, where_d, numRows);
                // eval(data, offsets, &d.mdata.datatypes, &tree[0], , , 0);
                hipDeviceSynchronize();
                hipError_t err = hipGetLastError();
                if (err != hipSuccess) {
                    printf("Error at %d: %s\n", __LINE__, hipGetErrorString(err));
                }
                numRows = d->read(data);
            }

            // Free all the data
            d->~Data();
            free(d);
            free(data);
            free(offsets);
            hipFree(data_d);
            hipFree(type_d);
            hipFree(where_d);
            hipFree(offsets_d);
        } else {
            // RETURN ALL ROWS
        }
    } else {
        fprintf(stderr, "Given string is not a valid SQL query.\n");
        fprintf(stderr, "%s (L%d:%d)\n",
                result->errorMsg(),
                result->errorLine(),
                result->errorColumn());
    }
    free(result);
}