#include "hip/hip_runtime.h"
//
// Created by gautam on 25/04/20.
//

#include "sql_select.cuh"

#define NUM_THREADS 512

__global__ void selectKernel(void *data, int rowSize, int *offset, int offsetSize, ColType *types, myExpr *exprs, int numRows) {
    int rowsPerBlock = (numRows + NUM_THREADS - 1) / NUM_THREADS;
    unsigned int start = rowsPerBlock * threadIdx.x;
    unsigned int end = rowsPerBlock * (threadIdx.x + 1);

    void *res;
    int resType = 1;
    void *row;
    bool flag;
    for (unsigned int i = start; i < end; i++) {
        row = (char *)data + i * rowSize;
        eval(row, offset, types, exprs, res, resType);
        if (i < numRows) {
            flag = false;
            if (resType == RESTYPE_INT) {
                flag = *(int *) res != 0;
            } else if (resType == RESTYPE_FLT) {
                flag = *(float *) res != 0;
            }
            free(res);
            if (!flag) continue;
            // Condition is satisfied, write code here
            printRowDevice(row, types, offsetSize);
        }
    }
}

__global__ void
joinKernel(void *left, void *right, void *join, int joinType, myExpr *joinExpr, int *offset, int offsetSize, ColType *types, int rowSizeL, int rowSizeR, int numRowsL, int numRowsR,
           unsigned int *numRowsRes) {
    if (threadIdx.x == 0) {
        *numRowsRes = 0;
    }
    __syncthreads();
    int rowsPerThread = (numRowsL * numRowsR + NUM_THREADS - 1) / NUM_THREADS;
    const unsigned start = rowsPerThread * threadIdx.x;
    const unsigned end = rowsPerThread * (threadIdx.x + 1);
    const int rowSizeRes = rowSizeL + rowSizeR;

    void *res;
    int resType = 0;
    void *row;
    bool flag;
    unsigned old;
    for (unsigned i = start; i < end; ++i) {
        // row i in join is obtained from i / numRowsR from left and i % numRowsR in right
        unsigned l = i / numRowsR, r = i % numRowsR;
        if (l >= numRowsL || r >= numRowsR) break;
        row = malloc(rowSizeRes);
        memcpy(row, left, rowSizeL);
        memcpy((char *) row + rowSizeL, right, rowSizeR);
        eval(row, offset, types, joinExpr, res, resType);
        flag = false;
        if (resType == RESTYPE_INT) {
            flag = *(int *)res == 0;
        } else if (resType == RESTYPE_FLT) {
            flag = *(float *)res == 0;
        }
        free(res);
        if (!flag) continue;
        // Add this row to new table
        old = atomicInc(numRowsRes, numRowsL * numRowsR);
        memcpy((char *) join + old * rowSizeRes, row, rowSizeRes);
    }
}


void sql_select::execute(std::string &query) {

    hsql::SQLParserResult *result;
    std::vector<std::string> columnNames;

    result = hsql::SQLParser::parseSQLString(query);
    columnNames = std::vector<std::string>();

    if(result->isValid()){
        const auto *stmt = (const hsql::SelectStatement *) result->getStatement(0);
        // Get column names
        for (hsql::Expr* expr : *stmt->selectList){
            switch (expr->type) {
                case hsql::kExprStar:
                    columnNames.emplace_back("*");
                    break;
                case hsql::kExprColumnRef:
                    columnNames.emplace_back(expr->name);
                    break;
                // case hsql::kExprTableColumnRef:
                // inprint(expr->table, expr->name, numIndent);
                // break;
                case hsql::kExprLiteralFloat:
                    columnNames.push_back(std::to_string(expr->fval));
                    break;
                case hsql::kExprLiteralInt:
                    columnNames.push_back(std::to_string(expr->ival));
                    break;
                case hsql::kExprLiteralString:
                    columnNames.emplace_back(expr->name);
                    break;
                // TODO: kExprFunctionRef (Distinct ?), kExprOperator (col1 + col2 ?)
                // case hsql::kExprFunctionRef:
                //     inprint(expr->name, numIndent);
                //     inprint(expr->expr->name, numIndent + 1);
                //     break;
                // case hsql::kExprOperator:
                //     printOperatorExpression(expr, numIndent);
                //     break;
                default:
                    fprintf(stderr, "Unrecognized expression type %d\n", expr->type);
                    return;
            }
        }
        // Get tables reference
        auto table = stmt->fromTable;
        Data *d;
        switch (table->type) {
            case hsql::kTableName:
                // inprint(table->name, numIndent);
                d = new Data(table->name);
                break;
            // case hsql::kTableSelect:
            //     // printSelectStatementInfo(table->select, numIndent);
            //     break;
            case hsql::kTableJoin: {
                //     // inprint("Join Table", numIndent);
                //     // inprint("Left", numIndent + 1);
                //     // printTableRefInfo(table->join->left, numIndent + 2);
                //     // inprint("Right", numIndent + 1);
                //     // printTableRefInfo(table->join->right, numIndent + 2);
                //     // inprint("Join Condition", numIndent + 1);
                //     // printExpression(table->join->condition, numIndent + 2);
                d = new Data(table->join->left->name, table->join->right->name);
                Data dL(table->join->left->name);
                dL.chunkSize = d->chunkSize;
                Data dR(table->join->right->name);
                dR.chunkSize = d->chunkSize;
                void *join = malloc(d->chunkSize * d->chunkSize * d->mdata.rowSize); // Upto n^2 rows can be stored
                void *dataL = malloc(d->chunkSize * dL.mdata.rowSize);
                void *dataR = malloc(d->chunkSize * dR.mdata.rowSize);
                int bytesReadL = dL.read(dataL), bytesReadR;
                // while (bytesReadL > 0) {
                //
                //     bytesReadR = dR.read(dataR);
                //     while (bytesReadR > 0) {
                //
                //     }
                // }
                break;
            }
            // case hsql::kTableCrossProduct:
            //     // for (TableRef* tbl : *table->list) printTableRefInfo(tbl, numIndent);
            //     break;
            default:
                printf("Will be handled later\n");
                return;
        }
        if (stmt->whereClause != nullptr) {
            // Get where
            std::vector<myExpr> tree;
            // printf("%s\n", tableNames[0].c_str());
            // Data d(tableNames[0]);

            auto expr = stmt->whereClause;
            exprToVec(expr, tree, d->mdata.columns);
            free(expr);

            int rowSize = d->mdata.rowSize;
            void *data = malloc(d->chunkSize * rowSize);
            void *data_d;
            int numCols = d->mdata.columns.size();
            ColType *type_d;
            hipSetDevice(0);
            hipDeviceReset();

            hipMalloc(&type_d, sizeof(ColType) * numCols);
            hipMemcpy(type_d, &d->mdata.datatypes[0], sizeof(ColType) * numCols, hipMemcpyHostToDevice);
            myExpr *where_d;
            hipMalloc(&where_d, sizeof(myExpr) * tree.size());
            hipMemcpy(where_d, &tree[0], sizeof(myExpr) * tree.size(), hipMemcpyHostToDevice);
            int *offsets = (int *) malloc(sizeof(int) * (numCols + 1));
            offsets[0] = 0;
            for (int i = 1; i <= numCols; i++) {
                offsets[i] = offsets[i - 1] + d->mdata.datatypes[i - 1].size;
            }
            int *offsets_d;
            hipMalloc(&offsets_d, sizeof(int) * (numCols + 1));
            hipMemcpy(offsets_d, offsets, sizeof(int) * (numCols + 1), hipMemcpyHostToDevice);
            int numRows = d->read(data);

            // printing data in table
            // utils::printMultiple(data, d.mdata.datatypes, d.mdata.rowSize, d.mdata.rowCount);

            hipMalloc(&data_d, d->chunkSize * rowSize);
            while (numRows > 0) {
                // printf("Inside\n");
                // fflush(stdout);
                hipMemcpy(data_d, data, rowSize * numRows, hipMemcpyHostToDevice);
                selectKernel<<<1, NUM_THREADS>>>(data_d, rowSize, offsets_d, numCols, type_d, where_d, numRows);
                // eval(data, offsets, &d.mdata.datatypes, &tree[0], , , 0);
                hipDeviceSynchronize();
                hipError_t err = hipGetLastError();
                if (err != hipSuccess) {
                    printf("Error at %d: %s\n", __LINE__, hipGetErrorString(err));
                }
                numRows = d->read(data);
            }

            // Free all the data
            free(data);
            free(offsets);
            hipFree(data_d);
            hipFree(type_d);
            hipFree(where_d);
            hipFree(offsets_d);
        } else {
            // RETURN ALL ROWS
        }
    } else {
        fprintf(stderr, "Given string is not a valid SQL query.\n");
        fprintf(stderr, "%s (L%d:%d)\n",
                result->errorMsg(),
                result->errorLine(),
                result->errorColumn());
    }
    free(result);
}