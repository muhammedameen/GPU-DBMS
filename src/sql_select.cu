#include "hip/hip_runtime.h"
//
// Created by gautam on 25/04/20.
//

#include "sql_select.cuh"

#define NUM_THREADS 512

__global__ void selectKernel(void *data, int rowSize, int *offset, int offsetSize, ColType *types, myExpr *exprs, int numRows, const int *dispCols, int numDispCols) {
    int rowsPerBlock = (numRows + NUM_THREADS - 1) / NUM_THREADS;
    unsigned int start = rowsPerBlock * threadIdx.x;
    unsigned int end = rowsPerBlock * (threadIdx.x + 1);

    void *res;
    int resType = 1;
    void *row;
    bool flag;
    for (unsigned int i = start; i < end; i++) {
        if (i < numRows) {
            row = (char *)data + i * rowSize;
            eval(row, offset, types, exprs, res, resType);
            flag = false;
            if (resType == RESTYPE_INT) {
                flag = *(int *) res != 0;
            } else if (resType == RESTYPE_FLT) {
                flag = *(float *) res != 0;
            }
            free(res);
            if (!flag) continue;
            // Condition is satisfied, write code here
            // printf("Row id: %d", i);
            printRowDevice(row, types, offsetSize, dispCols, numDispCols, offset);
        }
    }
}

__global__ void selectKernelRes(void *data, int rowSize, int *offset, int offsetSize, ColType *types, myExpr *exprs,
        int numRows, void *resData, unsigned int *top) {
    if (threadIdx.x == 0) {
        *top = 0;
    }
    __syncthreads();

    int rowsPerBlock = (numRows + NUM_THREADS - 1) / NUM_THREADS;
    unsigned int start = rowsPerBlock * threadIdx.x;
    unsigned int end = rowsPerBlock * (threadIdx.x + 1);

    void *res;
    int resType = 1;
    void *row;
    bool flag;
    unsigned int old;
    for (unsigned int i = start; i < end; i++) {
        if (i < numRows) {
            row = (char *)data + i * rowSize;
            eval(row, offset, types, exprs, res, resType);
            flag = false;
            if (resType == RESTYPE_INT) {
                flag = *(int *) res != 0;
            } else if (resType == RESTYPE_FLT) {
                flag = *(float *) res != 0;
            }
            free(res);
            if (!flag) continue;
            // Condition is satisfied, write code here
            // printRowDevice(row, types, offsetSize);
            old = atomicInc(top, numRows + 1);
            memcpy((char *) resData + old * rowSize, row, rowSize);
        }
    }
}

__global__ void
joinKernel(void *left, void *right, void *join, myExpr *joinExpr, int *offset, int numCols, ColType *types, int rowSizeL, int rowSizeR, int numRowsL, int numRowsR,
           unsigned int *numRowsRes, bool *matchedL) {
    if (threadIdx.x == 0) {
        *numRowsRes = 0;
    }
    __syncthreads();
    int rowsPerThread = (numRowsL * numRowsR + NUM_THREADS - 1) / NUM_THREADS;
    const unsigned start = rowsPerThread * threadIdx.x;
    const unsigned end = rowsPerThread * (threadIdx.x + 1);
    const int rowSizeRes = rowSizeL + rowSizeR;

    void *res;
    int resType = 0;
    void *row;
    bool flag;
    unsigned old;

    unsigned l_prev = numRowsL + 1;
    unsigned l, r;
    for (unsigned i = start; i < end; ++i) {
        // row i in join is obtained from i / numRowsR from left and i % numRowsR in right
        l = i / numRowsR;
        r = i % numRowsR;
        if (l >= numRowsL || r >= numRowsR) break;
        // printf("[%d, %d, (%d, %d)]\n", threadIdx.x, i, l, r);
        row = malloc(rowSizeRes);
        memcpy(row, (char *)left + l * rowSizeL, rowSizeL);
        memcpy((char *) row + rowSizeL, (char *)right + r * rowSizeR, rowSizeR);
        eval(row, offset, types, joinExpr, res, resType);
        flag = false;
        if (resType == RESTYPE_INT) {
            flag = *(int *)res != 0;
        } else if (resType == RESTYPE_FLT) {
            flag = *(float *)res != 0;
        }
        free(res);
        if (!flag) continue;
        if (l != l_prev) {
            matchedL[l] = true;
            l_prev = l;
        }
        old = atomicInc(numRowsRes, numRowsL * numRowsR);
        memcpy((char *) join + old * rowSizeRes, row, rowSizeRes);
        // printRowDevice(row, types, numCols);
    }
}

__global__ void
getLeft(void *data, bool *matched, int numRows, ColType *typesNew, const int numColsOld, const int numColsNew,
        const int rowSizeOld, const int rowSizeNew, void *resData, unsigned int *top) {
    if (threadIdx.x == 0) {
        *top = 0;
    }
    __syncthreads();
    int rowsPerBlock = (numRows + NUM_THREADS - 1) / NUM_THREADS;
    unsigned int start = rowsPerBlock * threadIdx.x;
    unsigned int end = rowsPerBlock * (threadIdx.x + 1);
    void *rowOld, *row, *cell;
    row = malloc(rowSizeNew);
    cell = (char *)row + rowSizeOld;
    for (int j = numColsOld; j < numColsNew; ++j) {
        switch (typesNew[j].type) {
            case TYPE_INT: {
                int *x = (int *) cell;
                *x = getNullInt();
                break;
            }
            case TYPE_FLOAT: {
                float *x = (float *) cell;
                *x = getNullFlt();
                break;
            }
            case TYPE_VARCHAR: {
                getNullStr((char *)cell, typesNew[j].size);
                break;
            }
            default:
                printf("Not implemented\n");
                return;
        }
        cell = (char *) cell + typesNew[j].size;
    }
    unsigned int old;
    for (int i = start; i < end; ++i) {
        if (i >= numRows) break;
        if (!matched[i]) {
            rowOld = (char *) data + rowSizeOld * i;
            memcpy(row, rowOld, rowSizeOld);
            // printRowDevice(row, typesNew, numColsNew);
            old = atomicInc(top, numRows + 1);
            memcpy((char *) resData + old * rowSizeNew, row, rowSizeNew);
        }
    }
    free(row);
}

void sql_select::execute(std::string &query) {
    hsql::SQLParserResult *result = hsql::SQLParser::parseSQLString(query);
    std::vector<int> columnNames;

    if(!result->isValid()) {
        utils::invalidQuery(result->errorMsg());
        return;
    }

    hipDeviceReset();

    const auto *stmt = (const hsql::SelectStatement *) result->getStatement(0);

    Data *d = getData(stmt->fromTable);

    for (hsql::Expr* expr : *stmt->selectList){
        switch (expr->type) {
            case hsql::kExprStar:
                // columnNames.emplace_back("*");
                columnNames.push_back(-1);
                break;
            case hsql::kExprColumnRef:
                // columnNames.emplace_back(expr->name);
                columnNames.push_back(d->mdata.colMap[expr->name]);
                break;
                // case hsql::kExprTableColumnRef:
                // inprint(expr->table, expr->name, numIndent);
                // break;
            case hsql::kExprLiteralFloat:
                // columnNames.push_back(std::to_string(expr->fval));
                columnNames.push_back(-2);
                break;
            case hsql::kExprLiteralInt:
                // columnNames.push_back(std::to_string(expr->ival));
                columnNames.push_back(-2);
                break;
            case hsql::kExprLiteralString:
                // columnNames.emplace_back(expr->name);
                columnNames.push_back(-2);
                break;
                // TODO: kExprFunctionRef (Distinct ?), kExprOperator (col1 + col2 ?)
                // case hsql::kExprFunctionRef:
                //     inprint(expr->name, numIndent);
                //     inprint(expr->expr->name, numIndent + 1);
                //     break;
                // case hsql::kExprOperator:
                //     printOperatorExpression(expr, numIndent);
                //     break;
            default:
                fprintf(stderr, "Unrecognized expression type %d\n", expr->type);
                return;
        }
    }


    std::vector<myExpr> whereExpr;
    if (stmt->whereClause != nullptr) {
        exprToVec(stmt->whereClause, whereExpr, d->mdata.columns, *d);
    } else {
        whereExpr.push_back(newExpr(CONSTANT_INT, (long) 1));
    }
    int rowSize = d->mdata.rowSize;
    void *data = malloc(d->chunkSize * rowSize);
    void *data_d;
    int numCols = d->mdata.columns.size();

    ColType *type_d;
    hipMalloc(&type_d, sizeof(ColType) * numCols);
    hipMemcpy(type_d, &d->mdata.datatypes[0], sizeof(ColType) * numCols, hipMemcpyHostToDevice);

    myExpr *where_d;
    hipMalloc(&where_d, sizeof(myExpr) * whereExpr.size());
    hipMemcpy(where_d, &whereExpr[0], sizeof(myExpr) * whereExpr.size(), hipMemcpyHostToDevice);

    std::vector<int> offsets(numCols + 1);
    offsets[0] = 0;
    for (int i = 1; i <= numCols; ++i) {
        offsets[i] = offsets[i - 1] + d->mdata.datatypes[i - 1].size;
    }
    int *offsets_d;
    hipMalloc(&offsets_d, sizeof(int) * (numCols + 1));
    hipMemcpy(offsets_d, &offsets[0], sizeof(int) * (numCols + 1), hipMemcpyHostToDevice);

    int *dispCols_d;
    hipMalloc(&dispCols_d, sizeof(int) * columnNames.size());
    hipMemcpy(dispCols_d, &columnNames[0], sizeof(int) * columnNames.size(), hipMemcpyHostToDevice);

    int rowsRead = d->read(data);
    // printf("HERE____________________%d\n", rowSize);
    // utils::printMultiple(data, d->mdata.datatypes, d->mdata.rowSize, d->mdata.rowCount);
    hipMalloc(&data_d, d->chunkSize * rowSize);
    while (rowsRead > 0) {
        hipMemcpy(data_d, data, rowSize * rowsRead, hipMemcpyHostToDevice);
        // hipMemcpy(data, data_d, rowSize * rowsRead, hipMemcpyDeviceToHost);
        // printf("HERE____________________%d\n", rowSize);
        // utils::printMultiple(data, d->mdata.datatypes, d->mdata.rowSize, d->mdata.rowCount);
        selectKernel<<<1, NUM_THREADS>>>(data_d, rowSize, offsets_d, numCols, type_d, where_d, rowsRead, dispCols_d,
                                         columnNames.size());
        rowsRead = d->read(data);
        hipDeviceSynchronize();
    }

    d->~Data();
    free(d);
    free(data);

    hipFree(data_d);
    hipFree(type_d);
    hipFree(where_d);
    hipFree(offsets_d);
}

Data *sql_select::getData(hsql::TableRef *fromTable) {
    switch (fromTable->type) {
        case hsql::kTableName:
            return new Data(fromTable->name);
        case hsql::kTableSelect:
            return selectData(fromTable->select);
        case hsql::kTableJoin: {
            Data *d, *dL, *dR;
            if (fromTable->join->type != hsql::kJoinRight) {
                dL = getData(fromTable->join->left);
                dR = getData(fromTable->join->right);
            } else {
                dL = getData(fromTable->join->right);
                dR = getData(fromTable->join->left);
            }
            d = new Data(dL, dR);
            dL->chunkSize = dR->chunkSize = d->chunkSize;

            std::vector<myExpr> joinCondition;
            exprToVec(fromTable->join->condition, joinCondition, d->mdata.columns, *d);
            myExpr *joinCondition_d;
            hipMalloc(&joinCondition_d, joinCondition.size() * sizeof(myExpr));
            hipMemcpy(joinCondition_d, &joinCondition[0], sizeof(myExpr) * joinCondition.size(),
                       hipMemcpyHostToDevice);

            std::vector<int> offsets(d->mdata.columns.size() + 1);
            offsets[0] = 0;
            for (int i = 1; i <= d->mdata.columns.size(); ++i) {
                offsets[i] = offsets[i - 1] + d->mdata.datatypes[i - 1].size;
            }
            int *offsets_d;
            hipMalloc(&offsets_d, sizeof(int) * (d->mdata.columns.size() + 1));
            hipMemcpy(offsets_d, &offsets[0], sizeof(int) * (d->mdata.columns.size() + 1), hipMemcpyHostToDevice);

            ColType *type_d;
            hipMalloc(&type_d, sizeof(ColType) * d->mdata.columns.size());
            hipMemcpy(type_d, &d->mdata.datatypes[0], sizeof(ColType) * d->mdata.columns.size(),
                       hipMemcpyHostToDevice);

            void *join = malloc(d->chunkSize * d->chunkSize * d->mdata.rowSize);
            void *dataL = malloc(d->chunkSize * dL->mdata.rowSize), *dataL_d;
            void *dataR = malloc(d->chunkSize * dR->mdata.rowSize), *dataR_d;
            void *join_d; // Upto n^2 rows can be stored
            hipMalloc(&join_d, d->chunkSize * d->chunkSize * d->mdata.rowSize);
            hipMalloc(&dataL_d, dL->chunkSize * dL->mdata.rowSize);
            hipMalloc(&dataR_d, dR->chunkSize * dR->mdata.rowSize);
            unsigned int numRowsJoin = 0;
            unsigned int *numRowsJoin_d;
            hipMalloc(&numRowsJoin_d, sizeof(unsigned int));

            std::vector<myExpr> whereClause;

            int rowsReadL = dL->read(dataL), rowsReadR;
            hipMemcpy(dataL_d, dataL, rowsReadL * dL->mdata.rowSize, hipMemcpyHostToDevice);

            bool *matched_d;
            hipMalloc(&matched_d, sizeof(bool) * dL->chunkSize);
            while (rowsReadL > 0) {
                hipMemset(matched_d, 0, sizeof(bool) * d->chunkSize);
                dR->restartRead();
                rowsReadR = dR->read(dataR);
                hipMemcpy(dataR_d, dataR, rowsReadR * dR->mdata.rowSize, hipMemcpyHostToDevice);
                while (rowsReadR > 0) {
                    joinKernel<<<1, NUM_THREADS>>>(dataL_d, dataR_d, join_d, joinCondition_d, offsets_d,
                                                   d->mdata.columns.size(),
                                                   type_d, dL->mdata.rowSize, dR->mdata.rowSize, rowsReadL, rowsReadR,
                                                   numRowsJoin_d, matched_d);
                    rowsReadR = dR->read(dataR);
                    hipDeviceSynchronize();

                    hipMemcpy(&numRowsJoin, numRowsJoin_d, sizeof(unsigned int), hipMemcpyDeviceToHost);
                    hipMemcpy(join, join_d, numRowsJoin * d->mdata.rowSize, hipMemcpyDeviceToHost);
                    d->write(join, numRowsJoin * d->mdata.rowSize);
                    fflush(stdout);
                    // selectKernel<<<1, NUM_THREADS>>>(join_d, d->mdata.rowSize, offsets_d, d->mdata.columns.size(),
                    //                                  type_d, whereClause_d, numRowsJoin);
                    hipDeviceSynchronize();
                    hipMemcpy(dataR_d, dataR, rowsReadR * dR->mdata.rowSize, hipMemcpyHostToDevice);
                }
                if (fromTable->join->type == hsql::kJoinLeft || fromTable->join->type == hsql::kJoinRight) {
                    getLeft<<<1, NUM_THREADS>>>(dataL_d, matched_d, rowsReadL, type_d, dL->mdata.columns.size(),
                                                d->mdata.columns.size(), dL->mdata.rowSize, d->mdata.rowSize, join_d,
                                                numRowsJoin_d);
                }
                rowsReadL = dL->read(dataL);
                hipDeviceSynchronize();
                if (fromTable->join->type == hsql::kJoinLeft || fromTable->join->type == hsql::kJoinRight) {
                    hipMemcpy(&numRowsJoin, numRowsJoin_d, sizeof(unsigned int), hipMemcpyDeviceToHost);
                    hipMemcpy(join, join_d, numRowsJoin * d->mdata.rowSize, hipMemcpyDeviceToHost);
                    d->write(join, numRowsJoin * d->mdata.rowSize);
                }
                hipMemcpy(dataL_d, dataL, rowsReadL * dL->mdata.rowSize, hipMemcpyHostToDevice);
            }
            dL->~Data();
            dR->~Data();
            free(dL);
            free(dR);
            free(dataL);
            free(dataR);
            free(join);

            hipFree(dataL_d);
            hipFree(dataR_d);
            hipFree(join_d);
            hipFree(joinCondition_d);
            hipFree(offsets_d);
            hipFree(type_d);
            hipFree(numRowsJoin_d);
            hipFree(matched_d);

            d->switchToRead();
            return d;
        }
        case hsql::kTableCrossProduct:
            utils::invalidQuery("Cross Product is not Implemented.");
            return nullptr;
    }
    return nullptr;
}

Data *sql_select::selectData(hsql::SelectStatement *stmt) {
    std::vector<std::string> columnNames;
    for (hsql::Expr* expr : *stmt->selectList){
        switch (expr->type) {
            case hsql::kExprStar:
                columnNames.emplace_back("*");
                break;
            case hsql::kExprColumnRef:
                columnNames.emplace_back(expr->name);
                break;
                // case hsql::kExprTableColumnRef:
                // inprint(expr->table, expr->name, numIndent);
                // break;
            case hsql::kExprLiteralFloat:
                columnNames.push_back(std::to_string(expr->fval));
                break;
            case hsql::kExprLiteralInt:
                columnNames.push_back(std::to_string(expr->ival));
                break;
            case hsql::kExprLiteralString:
                columnNames.emplace_back(expr->name);
                break;
                // TODO: kExprFunctionRef (Distinct ?), kExprOperator (col1 + col2 ?)
                // case hsql::kExprFunctionRef:
                //     inprint(expr->name, numIndent);
                //     inprint(expr->expr->name, numIndent + 1);
                //     break;
                // case hsql::kExprOperator:
                //     printOperatorExpression(expr, numIndent);
                //     break;
            default:
                fprintf(stderr, "Unrecognized expression type %d\n", expr->type);
                return nullptr;
        }
    }
    Data *d = getData(stmt->fromTable);
    Data *result = new Data(d);
    std::vector<myExpr> whereExpr;
    if (stmt->whereClause != nullptr) {
        exprToVec(stmt->whereClause, whereExpr, d->mdata.columns, *d);
    } else {
        whereExpr.push_back(newExpr(CONSTANT_INT, (long) 1));
    }
    int rowSize = d->mdata.rowSize;
    void *data = malloc(d->chunkSize * rowSize);
    void *resData = malloc(d->chunkSize * rowSize);
    void *data_d, *resData_d;
    int numCols = d->mdata.columns.size();

    ColType *type_d;
    hipMalloc(&type_d, sizeof(ColType) * numCols);
    hipMemcpy(type_d, &d->mdata.datatypes[0], sizeof(ColType) * numCols, hipMemcpyHostToDevice);

    myExpr *where_d;
    hipMalloc(&where_d, sizeof(myExpr) * whereExpr.size());
    hipMemcpy(where_d, &whereExpr[0], sizeof(myExpr) * whereExpr.size(), hipMemcpyHostToDevice);

    std::vector<int> offsets(numCols + 1);
    offsets[0] = 0;
    for (int i = 1; i <= numCols; ++i) {
        offsets[i] = offsets[i - 1] + d->mdata.datatypes[i - 1].size;
    }
    int *offsets_d;
    hipMalloc(&offsets_d, sizeof(int) * (numCols + 1));
    hipMemcpy(offsets_d, &offsets[0], sizeof(int) * (numCols + 1), hipMemcpyHostToDevice);

    int rowsRead = d->read(data);
    hipMalloc(&data_d, d->chunkSize * rowSize);
    hipMalloc(&resData_d, d->chunkSize * rowSize);
    unsigned int numMatches, *numMatches_d;
    hipMalloc(&numMatches_d, sizeof(unsigned int));
    while (rowsRead > 0) {
        hipMemcpy(data_d, data, rowSize * rowsRead, hipMemcpyHostToDevice);
        selectKernelRes<<<1, NUM_THREADS>>>(data_d, rowSize, offsets_d, numCols, type_d, where_d, rowsRead, resData_d,
                                            numMatches_d);
        rowsRead = d->read(data);
        hipDeviceSynchronize();
        hipMemcpy(&numMatches, numMatches_d, sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(resData, resData_d, rowSize * numMatches, hipMemcpyDeviceToHost);
        result->write(resData, rowSize * numMatches);
        // utils::printMultiple(resData, d->mdata.datatypes, rowSize, numMatches);
    }

    d->~Data();
    free(d);
    free(data);
    free(resData);

    hipFree(data_d);
    hipFree(resData_d);
    hipFree(type_d);
    hipFree(where_d);
    hipFree(offsets_d);
    result->switchToRead();
    result->restartRead();
    return result;
}

// void sql_select::execute(std::string &query) {
//
//     hsql::SQLParserResult *result;
//     std::vector<std::string> columnNames;
//
//     result = hsql::SQLParser::parseSQLString(query);
//     columnNames = std::vector<std::string>();
//
//     if(result->isValid()){
//         const auto *stmt = (const hsql::SelectStatement *) result->getStatement(0);
//         // Get column names
//         for (hsql::Expr* expr : *stmt->selectList){
//             switch (expr->type) {
//                 case hsql::kExprStar:
//                     columnNames.emplace_back("*");
//                     break;
//                 case hsql::kExprColumnRef:
//                     columnNames.emplace_back(expr->name);
//                     break;
//                 // case hsql::kExprTableColumnRef:
//                 // inprint(expr->table, expr->name, numIndent);
//                 // break;
//                 case hsql::kExprLiteralFloat:
//                     columnNames.push_back(std::to_string(expr->fval));
//                     break;
//                 case hsql::kExprLiteralInt:
//                     columnNames.push_back(std::to_string(expr->ival));
//                     break;
//                 case hsql::kExprLiteralString:
//                     columnNames.emplace_back(expr->name);
//                     break;
//                 // TODO: kExprFunctionRef (Distinct ?), kExprOperator (col1 + col2 ?)
//                 // case hsql::kExprFunctionRef:
//                 //     inprint(expr->name, numIndent);
//                 //     inprint(expr->expr->name, numIndent + 1);
//                 //     break;
//                 // case hsql::kExprOperator:
//                 //     printOperatorExpression(expr, numIndent);
//                 //     break;
//                 default:
//                     fprintf(stderr, "Unrecognized expression type %d\n", expr->type);
//                     return;
//             }
//         }
//         // Get tables reference
//         auto table = stmt->fromTable;
//         Data *d;
//         switch (table->type) {
//             case hsql::kTableName:
//                 // inprint(table->name, numIndent);
//                 d = new Data(table->name);
//                 break;
//             // case hsql::kTableSelect:
//             //     // printSelectStatementInfo(table->select, numIndent);
//             //     break;
//             case hsql::kTableJoin: {
//                 std::string leftTable, rightTable;
//                 if (table->join->type != hsql::kJoinRight) {
//                     d = new Data(table->join->left->name, table->join->right->name);
//                     leftTable = table->join->left->name;
//                     rightTable = table->join->right->name;
//                 } else {
//                     d = new Data(table->join->right->name, table->join->left->name);
//                     leftTable = table->join->right->name;
//                     rightTable = table->join->left->name;
//                 }
//                 Data dL(leftTable);
//                 dL.chunkSize = d->chunkSize;
//                 Data dR(rightTable);
//                 dR.chunkSize = d->chunkSize;
//
//                 std::vector<myExpr> joinCondition;
//                 exprToVec(table->join->condition, joinCondition, d->mdata.columns, *d);
//                 myExpr *joinCondition_d;
//                 hipMalloc(&joinCondition_d, joinCondition.size() * sizeof(myExpr));
//                 hipMemcpy(joinCondition_d, &joinCondition[0], sizeof(myExpr) * joinCondition.size(),
//                            hipMemcpyHostToDevice);
//
//                 std::vector<int> offsets(d->mdata.columns.size() + 1);
//                 offsets[0] = 0;
//                 for (int i = 1; i <= d->mdata.columns.size(); ++i) {
//                     offsets[i] = offsets[i - 1] + d->mdata.datatypes[i - 1].size;
//                 }
//                 int *offsets_d;
//                 hipMalloc(&offsets_d, sizeof(int) * (d->mdata.columns.size() + 1));
//                 hipMemcpy(offsets_d, &offsets[0], sizeof(int) * (d->mdata.columns.size() + 1), hipMemcpyHostToDevice);
//
//                 ColType *type_d;
//                 hipMalloc(&type_d, sizeof(ColType) * d->mdata.columns.size());
//                 hipMemcpy(type_d, &d->mdata.datatypes[0], sizeof(ColType) * d->mdata.columns.size(),
//                            hipMemcpyHostToDevice);
//
//                 void *join = malloc(d->chunkSize * d->chunkSize * d->mdata.rowSize);
//                 void *dataL = malloc(d->chunkSize * dL.mdata.rowSize), *dataL_d;
//                 void *dataR = malloc(d->chunkSize * dR.mdata.rowSize), *dataR_d;
//                 void *join_d; // Upto n^2 rows can be stored
//                 hipMalloc(&join_d, d->chunkSize * d->chunkSize * d->mdata.rowSize);
//                 hipMalloc(&dataL_d, dL.chunkSize * dL.mdata.rowSize);
//                 hipMalloc(&dataR_d, dR.chunkSize * dR.mdata.rowSize);
//                 unsigned int numRowsJoin = 0;
//                 unsigned int *numRowsJoin_d;
//                 hipMalloc(&numRowsJoin_d, sizeof(unsigned int));
//
//                 std::vector<myExpr> whereClause;
//
//                 int rowsReadL = dL.read(dataL), rowsReadR;
//                 hipMemcpy(dataL_d, dataL, rowsReadL * dL.mdata.rowSize, hipMemcpyHostToDevice);
//
//                 bool *matched_d;
//                 hipMalloc(&matched_d, sizeof(bool) * dL.chunkSize);
//                 while (rowsReadL > 0) {
//                     hipMemset(matched_d, 0, sizeof(bool) * dL.chunkSize);
//                     dR.restartRead();
//                     rowsReadR = dR.read(dataR);
//                     hipMemcpy(dataR_d, dataR, rowsReadR * dR.mdata.rowSize, hipMemcpyHostToDevice);
//                     while (rowsReadR > 0) {
//                         joinKernel<<<1, NUM_THREADS>>>(dataL_d, dataR_d, join_d, joinCondition_d, offsets_d,
//                                                        d->mdata.columns.size(),
//                                                        type_d, dL.mdata.rowSize, dR.mdata.rowSize, rowsReadL, rowsReadR,
//                                                        numRowsJoin_d, matched_d);
//                         rowsReadR = dR.read(dataR);
//                         hipDeviceSynchronize();
//
//                         hipMemcpy(&numRowsJoin, numRowsJoin_d, sizeof(unsigned int), hipMemcpyDeviceToHost);
//                         hipMemcpy(join, join_d, numRowsJoin * d->mdata.rowSize, hipMemcpyDeviceToHost);
//                         d->write(join, numRowsJoin * d->mdata.rowSize);
//                         fflush(stdout);
//                         // selectKernel<<<1, NUM_THREADS>>>(join_d, d->mdata.rowSize, offsets_d, d->mdata.columns.size(),
//                         //                                  type_d, whereClause_d, numRowsJoin);
//                         hipDeviceSynchronize();
//                         hipMemcpy(dataR_d, dataR, rowsReadR * dR.mdata.rowSize, hipMemcpyHostToDevice);
//                     }
//                     if (table->join->type == hsql::kJoinLeft || table->join->type == hsql::kJoinRight) {
//                         getLeft<<<1, NUM_THREADS>>>(dataL_d, matched_d, rowsReadL, type_d, dL.mdata.columns.size(),
//                                                       d->mdata.columns.size(), dL.mdata.rowSize, d->mdata.rowSize);
//
//                     }
//                     rowsReadL = dL.read(dataL);
//                     hipDeviceSynchronize();
//                     hipMemcpy(dataL_d, dataL, rowsReadL * dL.mdata.rowSize, hipMemcpyHostToDevice);
//                 }
//
//                 myExpr *whereClause_d;
//                 if (stmt->whereClause != nullptr) {
//                     exprToVec(stmt->whereClause, whereClause, d->mdata.columns, *d);
//                     hipMalloc(&whereClause_d, sizeof(myExpr) * whereClause.size());
//                     hipMemcpy(whereClause_d, &whereClause[0], sizeof(myExpr) * whereClause.size(),
//                                hipMemcpyHostToDevice);
//                 }
//
//                 // change chunk size before select
//                 // d->chunkSize = 500 * 1024 * 1024 / d->mdata.rowSize;
//                 // if chunksize is changed, join and join_d might need to be reallocated
//                 d->chunkSize *= d->chunkSize;
//
//                 // printf("____________________________________________________\n");
//                 d->switchToRead();
//                 int numRowsRead;
//                 numRowsRead = d->read(join);
//                 while (numRowsRead > 0) {
//                     hipMemcpy(join_d, join, numRowsRead * d->mdata.rowSize, hipMemcpyHostToDevice);
//                     selectKernel<<<1, NUM_THREADS>>>(join_d, d->mdata.rowSize, offsets_d, d->mdata.columns.size(),
//                                                      type_d, whereClause_d, numRowsRead);
//                     hipDeviceSynchronize();
//                     numRowsRead = d->read(join);
//                 }
//
//                 d->~Data();
//                 free(d);
//                 free(dataL);
//                 free(dataR);
//
//                 hipFree(dataL_d);
//                 hipFree(dataR_d);
//                 hipFree(join_d);
//                 hipFree(joinCondition_d);
//                 hipFree(offsets_d);
//                 hipFree(type_d);
//                 hipFree(numRowsJoin_d);
//                 hipFree(whereClause_d);
//                 hipFree(offsets_d);
//                 hipFree(matched_d);
//                 return;
//                 break;
//             }
//             // case hsql::kTableCrossProduct:
//             //     // for (TableRef* tbl : *table->list) printTableRefInfo(tbl, numIndent);
//             //     break;
//             default:
//                 printf("Will be handled later\n");
//                 return;
//         }
//         if (stmt->whereClause != nullptr) {
//             // Get where
//             std::vector<myExpr> tree;
//
//             auto expr = stmt->whereClause;
//             exprToVec(expr, tree, d->mdata.columns, *d);
//             free(expr);
//
//             int rowSize = d->mdata.rowSize;
//             void *data = malloc(d->chunkSize * rowSize);
//             void *data_d;
//             int numCols = d->mdata.columns.size();
//             hipSetDevice(0);
//             hipDeviceReset();
//
//             ColType *type_d;
//             hipMalloc(&type_d, sizeof(ColType) * numCols);
//             hipMemcpy(type_d, &d->mdata.datatypes[0], sizeof(ColType) * numCols, hipMemcpyHostToDevice);
//             myExpr *where_d;
//             hipMalloc(&where_d, sizeof(myExpr) * tree.size());
//             hipMemcpy(where_d, &tree[0], sizeof(myExpr) * tree.size(), hipMemcpyHostToDevice);
//             int *offsets = (int *) malloc(sizeof(int) * (numCols + 1));
//             offsets[0] = 0;
//             for (int i = 1; i <= numCols; i++) {
//                 offsets[i] = offsets[i - 1] + d->mdata.datatypes[i - 1].size;
//             }
//             int *offsets_d;
//             hipMalloc(&offsets_d, sizeof(int) * (numCols + 1));
//             hipMemcpy(offsets_d, offsets, sizeof(int) * (numCols + 1), hipMemcpyHostToDevice);
//             int numRows = d->read(data);
//
//             // printing data in table
//             // utils::printMultiple(data, d.mdata.datatypes, d.mdata.rowSize, d.mdata.rowCount);
//
//             hipMalloc(&data_d, d->chunkSize * rowSize);
//             while (numRows > 0) {
//                 hipMemcpy(data_d, data, rowSize * numRows, hipMemcpyHostToDevice);
//                 selectKernel<<<1, NUM_THREADS>>>(data_d, rowSize, offsets_d, numCols, type_d, where_d, numRows);
//                 numRows = d->read(data);
//                 hipDeviceSynchronize();
//             }
//
//             // Free all the data
//             d->~Data();
//             free(d);
//             free(data);
//             free(offsets);
//             hipFree(data_d);
//             hipFree(type_d);
//             hipFree(where_d);
//             hipFree(offsets_d);
//         } else {
//             // RETURN ALL ROWS
//         }
//     } else {
//         fprintf(stderr, "Given string is not a valid SQL query.\n");
//         fprintf(stderr, "%s (L%d:%d)\n",
//                 result->errorMsg(),
//                 result->errorLine(),
//                 result->errorColumn());
//     }
//     free(result);
// }