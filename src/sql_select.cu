#include "hip/hip_runtime.h"
//
// Created by gautam on 25/04/20.
//

#include "sql_select.cuh"

__global__ void selectKernel(void *data, int rowSize, int *offset, int offsetSize, ColType *types, whereExpr *exprs) {
    void *res;
    int resType = 0;
    eval(data, rowSize, offset, offsetSize, types, exprs, 0, res, resType);
    if (resType == RESTYPE_INT) {
        int *x = (int *) res;
        printf("Value of expression is : %d\n", *x);
    } else {
        printf("Res Type is : %d", resType);
    }
}

void sql_select::execute(std::string &query) {

    hsql::SQLParserResult *result;
    std::vector<std::string> columnNames;
    std::vector<std::string> tableNames;

    result = hsql::SQLParser::parseSQLString(query);
    columnNames = std::vector<std::string>();
    tableNames = std::vector<std::string>();

    if(result->isValid()){
        const auto *stmt = (const hsql::SelectStatement *) result->getStatement(0);
        // hsql::printSelectStatementInfo(stmt, 1);
        // Get column names
        for (hsql::Expr* expr : *stmt->selectList){
            switch (expr->type) {
                case hsql::kExprStar:
                    columnNames.emplace_back("*");
                    break;
                case hsql::kExprColumnRef:
                    columnNames.emplace_back(expr->name);
                    break;
                // case hsql::kExprTableColumnRef:
                // inprint(expr->table, expr->name, numIndent);
                // break;
                case hsql::kExprLiteralFloat:
                    columnNames.push_back(std::to_string(expr->fval));
                    break;
                case hsql::kExprLiteralInt:
                    columnNames.push_back(std::to_string(expr->ival));
                    break;
                case hsql::kExprLiteralString:
                    columnNames.emplace_back(expr->name);
                    break;
                // TODO: kExprFunctionRef (Distinct ?), kExprOperator (col1 + col2 ?)
                // case hsql::kExprFunctionRef:
                //     inprint(expr->name, numIndent);
                //     inprint(expr->expr->name, numIndent + 1);
                //     break;
                // case hsql::kExprOperator:
                //     printOperatorExpression(expr, numIndent);
                //     break;
                default:
                    fprintf(stderr, "Unrecognized expression type %d\n", expr->type);
                    return;
            }
        }
        // Get tables reference
        auto table = stmt->fromTable;
        switch (table->type) {
            case hsql::kTableName:
                // inprint(table->name, numIndent);
                tableNames.emplace_back(table->name);
                break;
            // case hsql::kTableSelect:
            //     // printSelectStatementInfo(table->select, numIndent);
            //     break;
            // case hsql::kTableJoin:
            //     // inprint("Join Table", numIndent);
            //     // inprint("Left", numIndent + 1);
            //     // printTableRefInfo(table->join->left, numIndent + 2);
            //     // inprint("Right", numIndent + 1);
            //     // printTableRefInfo(table->join->right, numIndent + 2);
            //     // inprint("Join Condition", numIndent + 1);
            //     // printExpression(table->join->condition, numIndent + 2);
            //     break;
            // case hsql::kTableCrossProduct:
            //     // for (TableRef* tbl : *table->list) printTableRefInfo(tbl, numIndent);
            //     break;
            default:
                printf("Will be handled later\n");
                return;
        }
        if (stmt->whereClause != nullptr) {
            // Get where
            std::vector<whereExpr> tree;
            printf("%s\n", tableNames[0].c_str());
            Data d(tableNames[0]);

            auto expr = stmt->whereClause;
            exprToVec(expr, tree, d.mdata.columns);
            free(expr);

            int rowSize = d.mdata.rowSize;
            void *data = malloc(d.chunkSize);
            void *data_d;
            int numCols = d.mdata.columns.size();
            ColType *type_d;
            hipMalloc(&type_d, sizeof(ColType) * numCols);
            hipMemcpy(type_d, &d.mdata.datatypes[0], sizeof(ColType) * numCols, hipMemcpyHostToDevice);
            whereExpr *where_d;
            hipMalloc(&where_d, sizeof(whereExpr) * tree.size());
            hipMemcpy(where_d, &tree[0], sizeof(whereExpr) * tree.size(), hipMemcpyHostToDevice);
            int *offsets = (int *) malloc(sizeof(int) * (numCols + 1));
            offsets[0] = d.mdata.datatypes[0].size;
            for (int i = 1; i <= numCols; i++) {
                offsets[i] = offsets[i - 1] + d.mdata.datatypes[i].size;
            }
            int *offsets_d;
            hipMalloc(&offsets_d, sizeof(int) * (numCols + 1));
            hipMemcpy(offsets_d, offsets, sizeof(int) * (numCols + 1), hipMemcpyHostToDevice);
            int numRows = d.read(data);

            while (numRows > 0) {
                printf("Inside\n");
                fflush(stdout);
                hipMalloc(&data_d, rowSize * numRows);
                hipMemcpy(data_d, data, rowSize * numRows, hipMemcpyHostToDevice);
                selectKernel<<<1, 512>>>(data_d, rowSize, offsets_d, numCols, type_d, where_d);
                hipDeviceSynchronize();
                hipError_t err = hipGetLastError();
                printf("Error at %d: %s\n", __LINE__, hipGetErrorString(err));
                numRows = d.read(data);
            }

            // FOR DEBUGGING
            // for (auto leaf : tree) {
            //     printf("TYPE: %d, ival: %d, fval: %f, sval: %s, left: %d, right: %d\n", leaf.type, leaf.iVal, leaf.fVal,
            //            leaf.sVal, leaf.childLeft, leaf.childRight);
            // }

            // TEST EVAL
            // ColType type[] = {newColType("int"), newColType("varchar(7)"), newColType("int"), newColType("float")};
            // int start[] = {0, 4, 12, 16, 20};
            // int end[] = {4, 12, 16, 20};
            // void *row = malloc(20 * sizeof(char));
            // int r1 = 5;
            // char r2[8] = "ab";
            // int r3 = 10;
            // float r4 = 0.05f;
            // memcpy((char *) row + start[0], &r1, end[0] - start[0]);
            // memcpy((char *) row + start[1], r2, end[1] - start[1]);
            // memcpy((char *) row + start[2], &r3, end[2] - start[2]);
            // memcpy((char *) row + start[3], &r4, end[3] - start[3]);
            //
            // Data *a = new Data("persons");
            // // a.write(row, 20);
            // free(row);
            // row = malloc(20 * sizeof(char));
            // a->mdata.rowCount = 1;
            // a->read(row);
            //
            //
            // // Revserse memcpy
            // memcpy(&r1, (char *) row + start[0], end[0] - start[0]);
            // memcpy(r2, (char *) row + start[1], end[1] - start[1]);
            // memcpy(&r3, (char *) row + start[2], end[2] - start[2]);
            // memcpy(&r4, (char *) row + start[3], end[3] - start[3]);
            // printf("R1: %d, R2: %s, R3:%d, R4:%f\n", r1, r2, r3, r4);
            //
            // void *row_d;
            // hipMalloc(&row_d, 20);
            // hipMemcpy(row_d, row, 20, hipMemcpyHostToDevice);
            //
            // int *offset_d;
            // hipMalloc(&offset_d, 5 * sizeof(int));
            // hipMemcpy(offset_d, start, 5 * sizeof(int), hipMemcpyHostToDevice);
            //
            // char *colNames_d;
            // int *colPos_d;
            // hipMalloc(&colNames_d, sizeof(char *) * 4 * 100);
            // hipMalloc(&colPos_d, sizeof(int) * 4);
            // int colPos[] = {0, 3, 6, 9};
            // for (int i = 0; i < 4; i++) {
            //     hipMemcpy(colNames_d + 3 * i, colNames[i], sizeof(char) * 3, hipMemcpyHostToDevice);
            // }
            // hipMemcpy(colPos_d, colPos, sizeof(int) * 4, hipMemcpyHostToDevice);
            //
            // ColType *types_d;
            // hipMalloc(&types_d, sizeof(ColType) * 4);
            // hipMemcpy(types_d, type, sizeof(ColType) * 4, hipMemcpyHostToDevice);
            //
            // whereExpr *whereClause;
            // hipMalloc(&whereClause, sizeof(whereExpr) * tree.size());
            // hipMemcpy(whereClause, &tree[0], sizeof(whereExpr) * tree.size(), hipMemcpyHostToDevice);

            // selectKernel<<<1, 1>>>(row, 20, offset_d, 4, colNames_d, colPos_d, types_d, whereClause);
            // hipDeviceSynchronize();
            // hipError_t err = hipGetLastError();
            // printf("Error at %d: %s\n", __LINE__, hipGetErrorString(err));
            // eval(row, 20, start, 4, colNames, type, &tree[0], 0, res, resType);
            // if (resType == RESTYPE_INT) {
            //     int *x = (int *) res;
            //     printf("Value of expression is : %d\n", *x);
            // }
            // END TEST EVAL
        } else {
            // RETURN ALL ROWS
        }
    } else {
        fprintf(stderr, "Given string is not a valid SQL query.\n");
        fprintf(stderr, "%s (L%d:%d)\n",
                result->errorMsg(),
                result->errorLine(),
                result->errorColumn());
    }
    free(result);
}