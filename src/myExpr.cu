#include "hip/hip_runtime.h"
//
// Created by gautam on 28/04/20.
//

#include <float.h>
#include "myExpr.cuh"

#define NUM_THREADS 512


myExpr newExpr(myExprType type, long intVal) {
    myExpr expr;
    expr.type = type;
    expr.iVal = (int)intVal;
    expr.fVal = 0.0f;
    expr.sVal[0] = 0;
    expr.childLeft = -1;
    expr.childRight = -1;
    return expr;
}

myExpr newExpr(myExprType type, float fVal){
    myExpr expr;
    expr.type = type;
    expr.iVal = 0;
    expr.fVal = fVal;
    expr.sVal[0] = 0;
    expr.childLeft = -1;
    expr.childRight = -1;
    return expr;
}

myExpr newExpr(myExprType type, char *sVal){
    myExpr expr;
    expr.type = type;
    expr.iVal = 0;
    expr.fVal = 0.0f;
    // expr->sVal = new char[strlen(sVal) + 1];
    stpcpy(expr.sVal, sVal);
    expr.childLeft = -1;
    expr.childRight = -1;
    return expr;
}

myExpr newExpr(myExprType type){
    myExpr expr;
    expr.type = type;
    expr.iVal = 0;
    expr.fVal = 0.0f;
    expr.sVal[0] = 0;
    expr.childLeft = -1;
    expr.childRight = -1;
    return expr;
}

void freeExpr(myExpr *expr){
    free(expr);
}

__device__ float atomicMax(float* address, float val)
{
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
                          __float_as_int(::fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

__device__ float atomicMin(float* address, float val)
{
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
                          __float_as_int(::fminf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

__global__ void minKernel(void *data, const int colPos, const int rowSize, const int numRows, int *min) {
    int rowsPerBlock = (numRows + NUM_THREADS - 1) / NUM_THREADS;
    unsigned int start = rowsPerBlock * threadIdx.x;
    unsigned int end = rowsPerBlock * (threadIdx.x + 1);
    int threadMin = *min;
    int *currVal;
    for (unsigned int i = start; i < end; ++i) {
        if (i >= numRows) break;
        currVal = (int *)((char *)data + i * rowSize + colPos);
        threadMin = threadMin < *currVal ? threadMin : *currVal;
    }
    atomicMin(min, threadMin);
}

__global__ void minKernel(void *data, const int colPos, const int rowSize, const int numRows, float *min) {
    int rowsPerBlock = (numRows + NUM_THREADS - 1) / NUM_THREADS;
    unsigned int start = rowsPerBlock * threadIdx.x;
    unsigned int end = rowsPerBlock * (threadIdx.x + 1);
    float threadMin = *min;
    float *currVal;
    for (unsigned int i = start; i < end; ++i) {
        if (i >= numRows) break;
        currVal = (float *)((char *)data + i * rowSize + colPos);
        threadMin = threadMin < *currVal ? threadMin : *currVal;
    }
    atomicMin(min, threadMin);
}

__global__ void maxKernel(void *data, const int colPos, const int rowSize, const int numRows, int *max) {
    int rowsPerBlock = (numRows + NUM_THREADS - 1) / NUM_THREADS;
    unsigned int start = rowsPerBlock * threadIdx.x;
    unsigned int end = rowsPerBlock * (threadIdx.x + 1);
    int threadMax = *max;
    int *currVal;
    for (unsigned int i = start; i < end; ++i) {
        if (i >= numRows) break;
        currVal = (int *)((char *)data + i * rowSize + colPos);
        threadMax = threadMax > *currVal ? threadMax : *currVal;
    }
    atomicMax(max, threadMax);
}

__global__ void maxKernel(void *data, const int colPos, const int rowSize, const int numRows, float *max) {
    int rowsPerBlock = (numRows + NUM_THREADS - 1) / NUM_THREADS;
    unsigned int start = rowsPerBlock * threadIdx.x;
    unsigned int end = rowsPerBlock * (threadIdx.x + 1);
    float threadMax = *max;
    float *currVal;
    for (unsigned int i = start; i < end; ++i) {
        if (i >= numRows) break;
        currVal = (float *)((char *)data + i * rowSize + colPos);
        threadMax = threadMax > *currVal ? threadMax : *currVal;
    }
    atomicMax(max, threadMax);
}

__global__ void sumKernel(void *data, const int colPos, const int rowSize, const int numRows, int *sum) {
    int rowsPerBlock = (numRows + NUM_THREADS - 1) / NUM_THREADS;
    unsigned int start = rowsPerBlock * threadIdx.x;
    unsigned int end = rowsPerBlock * (threadIdx.x + 1);
    int threadSum = *sum;
    int *currVal;
    for (unsigned int i = start; i < end; ++i) {
        if (i >= numRows) break;
        currVal = (int *)((char *)data + i * rowSize + colPos);
        threadSum += *currVal;
    }
    atomicAdd(sum, threadSum);
}

__global__ void sumKernel(void *data, const int colPos, const int rowSize, const int numRows, float *sum) {
    int rowsPerBlock = (numRows + NUM_THREADS - 1) / NUM_THREADS;
    unsigned int start = rowsPerBlock * threadIdx.x;
    unsigned int end = rowsPerBlock * (threadIdx.x + 1);
    float threadMax = *sum;
    float *currVal;
    for (unsigned int i = start; i < end; ++i) {
        if (i >= numRows) break;
        currVal = (float *)((char *)data + i * rowSize + colPos);
        threadMax += *currVal;
    }
    atomicAdd(sum, threadMax);
}

void exprToVec(hsql::Expr *expr, std::vector<myExpr> &vector, const std::vector<std::string>& colNames, Data &d) {
    switch (expr->type) {
        case hsql::kExprLiteralFloat:
            vector.push_back(newExpr(CONSTANT_FLT, expr->fval));
            break;
        case hsql::kExprLiteralString:
            vector.push_back(newExpr(CONSTANT_STR, expr->name));
            break;
        case hsql::kExprLiteralInt:
            vector.push_back(newExpr(CONSTANT_INT, expr->ival));
            break;
        case hsql::kExprStar:
            printf("Why is there a `*` here?");
            break;
        case hsql::kExprPlaceholder:
            printf("What is this?");
            break;
        case hsql::kExprColumnRef:{
            int i;
            for (i = 0; i < colNames.size(); i++) {
                if (colNames[i] == expr->name) break;
            }
            vector.push_back(newExpr(COL_NAME, (long)i));
            break;
        }
        case hsql::kExprFunctionRef: {
            // printf("%s\n", expr->name);
            int oldChunkSize = d.chunkSize;
            d.chunkSize *= 10;
            void *data = malloc(d.chunkSize * d.mdata.rowSize);
            void *data_d;
            hipMalloc(&data_d, d.chunkSize * d.mdata.rowSize);
            int rowsRead = d.read(data);
            hipMemcpy(data_d, data, rowsRead * d.mdata.rowSize, hipMemcpyHostToDevice);
            std::string colName = expr->exprList->at(0)->name;
            // printf("Col for agg function is %s:%d\n", colName.c_str(), d.mdata.colMap[colName]);
            fflush(stdout);
            int colPos = 0;
            int resType = TYPE_INT;

            for (int i = 0; i < colNames.size(); ++i) {
                if (colNames[i] == colName) {
                    resType = d.mdata.datatypes[i].type;
                    break;
                }
                colPos += d.mdata.datatypes[i].size;
            }

            if (strcmp(expr->name, "min") == 0) {
                if (resType == TYPE_INT) {
                    int min_h = INT_MAX;
                    int *min;
                    hipMalloc(&min, sizeof(int));
                    hipMemcpy(min, &min_h, sizeof(int), hipMemcpyHostToDevice);
                    while (rowsRead > 0) {
                        minKernel<<<1, NUM_THREADS>>>(data_d, colPos, d.mdata.rowSize, rowsRead, min);
                        rowsRead = d.read(data);
                        hipDeviceSynchronize();
                        hipMemcpy(data_d, data, rowsRead * d.mdata.rowSize, hipMemcpyHostToDevice);
                    }
                    hipMemcpy(&min_h, min, sizeof(int), hipMemcpyDeviceToHost);
                    hipFree(min);
                    printf("Min value is: %d\n", min_h);
                    vector.push_back(newExpr(CONSTANT_INT, (long) min_h));
                } else if (resType == TYPE_FLOAT) {
                    float min_h = FLT_MAX;
                    float *min;
                    hipMalloc(&min, sizeof(float));
                    hipMemcpy(min, &min_h, sizeof(float), hipMemcpyHostToDevice);
                    while (rowsRead > 0) {
                        minKernel<<<1, NUM_THREADS>>>(data_d, colPos, d.mdata.rowSize, rowsRead, min);
                        rowsRead = d.read(data);
                        hipDeviceSynchronize();
                        hipMemcpy(data_d, data, rowsRead * d.mdata.rowSize, hipMemcpyHostToDevice);
                    }
                    hipMemcpy(&min_h, min, sizeof(float), hipMemcpyDeviceToHost);
                    hipFree(min);
                    vector.push_back(newExpr(CONSTANT_FLT, min_h));
                }
            } else if (strcmp(expr->name, "max") == 0) {
                if (resType == TYPE_INT) {
                    int max_h = INT_MAX;
                    int *max;
                    hipMalloc(&max, sizeof(int));
                    hipMemcpy(max, &max_h, sizeof(int), hipMemcpyHostToDevice);
                    while (rowsRead > 0) {
                        maxKernel<<<1, NUM_THREADS>>>(data_d, colPos, d.mdata.rowSize, rowsRead, max);
                        rowsRead = d.read(data);
                        hipDeviceSynchronize();
                        hipMemcpy(data_d, data, rowsRead * d.mdata.rowSize, hipMemcpyHostToDevice);
                    }
                    hipMemcpy(&max_h, max, sizeof(int), hipMemcpyDeviceToHost);
                    hipFree(max);
                    printf("Max value is: %d\n", max_h);
                    vector.push_back(newExpr(CONSTANT_INT, (long) max_h));
                } else if (resType == TYPE_FLOAT) {
                    float max_h = FLT_MAX;
                    float *max;
                    hipMalloc(&max, sizeof(float));
                    hipMemcpy(max, &max_h, sizeof(float), hipMemcpyHostToDevice);
                    while (rowsRead > 0) {
                        maxKernel<<<1, NUM_THREADS>>>(data_d, colPos, d.mdata.rowSize, rowsRead, max);
                        rowsRead = d.read(data);
                        hipDeviceSynchronize();
                        hipMemcpy(data_d, data, rowsRead * d.mdata.rowSize, hipMemcpyHostToDevice);
                    }
                    hipMemcpy(&max_h, max, sizeof(float), hipMemcpyDeviceToHost);
                    hipFree(max);
                    vector.push_back(newExpr(CONSTANT_FLT, max_h));
                }
            } else if (strcmp(expr->name, "sum") == 0) {
                if (resType == TYPE_INT) {
                    int sum_h = 0;
                    int *sum;
                    hipMalloc(&sum, sizeof(int));
                    hipMemcpy(sum, &sum_h, sizeof(int), hipMemcpyHostToDevice);
                    while (rowsRead > 0) {
                        sumKernel<<<1, NUM_THREADS>>>(data_d, colPos, d.mdata.rowSize, rowsRead, sum);
                        rowsRead = d.read(data);
                        hipDeviceSynchronize();
                        hipMemcpy(data_d, data, rowsRead * d.mdata.rowSize, hipMemcpyHostToDevice);
                    }
                    hipMemcpy(&sum_h, sum, sizeof(int), hipMemcpyDeviceToHost);
                    hipFree(sum);
                    printf("Sum value is: %d\n", sum_h);
                    vector.push_back(newExpr(CONSTANT_INT, (long) sum_h));
                } else if (resType == TYPE_FLOAT) {
                    float sum_h = FLT_MAX;
                    float *sum;
                    hipMalloc(&sum, sizeof(float));
                    hipMemcpy(sum, &sum_h, sizeof(float), hipMemcpyHostToDevice);
                    while (rowsRead > 0) {
                        sumKernel<<<1, NUM_THREADS>>>(data_d, colPos, d.mdata.rowSize, rowsRead, sum);
                        rowsRead = d.read(data);
                        hipDeviceSynchronize();
                        hipMemcpy(data_d, data, rowsRead * d.mdata.rowSize, hipMemcpyHostToDevice);
                    }
                    hipMemcpy(&sum_h, sum, sizeof(float), hipMemcpyDeviceToHost);
                    hipFree(sum);
                    vector.push_back(newExpr(CONSTANT_FLT, sum_h));
                }
            } else if (strcmp(expr->name, "avg") == 0) {
                if (resType == TYPE_INT) {
                    int sum_h = 0;
                    int *sum;
                    hipMalloc(&sum, sizeof(int));
                    hipMemcpy(sum, &sum_h, sizeof(int), hipMemcpyHostToDevice);
                    while (rowsRead > 0) {
                        sumKernel<<<1, NUM_THREADS>>>(data_d, colPos, d.mdata.rowSize, rowsRead, sum);
                        rowsRead = d.read(data);
                        hipDeviceSynchronize();
                        hipMemcpy(data_d, data, rowsRead * d.mdata.rowSize, hipMemcpyHostToDevice);
                    }
                    hipMemcpy(&sum_h, sum, sizeof(int), hipMemcpyDeviceToHost);
                    hipFree(sum);
                    printf("Sum value is: %d\n", sum_h);
                    vector.push_back(newExpr(CONSTANT_INT, (long) sum_h));
                } else if (resType == TYPE_FLOAT) {
                    float sum_h = FLT_MAX;
                    float *sum;
                    hipMalloc(&sum, sizeof(float));
                    hipMemcpy(sum, &sum_h, sizeof(float), hipMemcpyHostToDevice);
                    while (rowsRead > 0) {
                        sumKernel<<<1, NUM_THREADS>>>(data_d, colPos, d.mdata.rowSize, rowsRead, sum);
                        rowsRead = d.read(data);
                        hipDeviceSynchronize();
                        hipMemcpy(data_d, data, rowsRead * d.mdata.rowSize, hipMemcpyHostToDevice);
                    }
                    hipMemcpy(&sum_h, sum, sizeof(float), hipMemcpyDeviceToHost);
                    hipFree(sum);
                    vector.push_back(newExpr(CONSTANT_FLT, sum_h));
                }
            } else if (strcmp(expr->name, "count") == 0) {

            }
            d.chunkSize = oldChunkSize;
            d.restartRead();
            free(data);
            hipFree(data_d);
            break;
        }
        case hsql::kExprOperator: {
            myExpr *temp = newExpr(getOpType(expr->opType, expr->opChar));
            vector.push_back(*temp);
            int curr = (int)vector.size() - 1;
            vector[curr].childLeft = vector.size();
            exprToVec(expr->expr, vector, colNames, d);
            if (expr->expr2 != nullptr) {
                vector[curr].childRight = vector.size();
                exprToVec(expr->expr2, vector, colNames, d);
            }
            break;
        }
        case hsql::kExprSelect:
            printf("Not yet implemented");
            break;
    }
}

myExprType getOpType(hsql::Expr::OperatorType type, char opChar) {
    // TODO: Change Error to correct Constants
    switch (type) {
        case hsql::Expr::NONE:
            return CONSTANT_ERR;
        case hsql::Expr::BETWEEN:
            return CONSTANT_ERR;
        case hsql::Expr::CASE:
            return CONSTANT_ERR;
        case hsql::Expr::SIMPLE_OP:
            switch (opChar) {
                case '+':
                    return OPERATOR_PL;
                case '-':
                    return OPERATOR_MI;
                case '*':
                    return OPERATOR_MU;
                case '/':
                    return OPERATOR_DI;
                case '%':
                    return OPERATOR_MO;
                case '=':
                    return OPERATOR_EQ;
                case '<':
                    return OPERATOR_LT;
                case '>':
                    return OPERATOR_GT;
                default:
                    return CONSTANT_ERR;
            }
        case hsql::Expr::NOT_EQUALS:
            return OPERATOR_NE;
        case hsql::Expr::LESS_EQ:
            return OPERATOR_LE;
        case hsql::Expr::GREATER_EQ:
            return OPERATOR_GE;
        case hsql::Expr::LIKE:
            return CONSTANT_ERR;
        case hsql::Expr::NOT_LIKE:
            return CONSTANT_ERR;
        case hsql::Expr::AND:
            return OPERATOR_AND;
        case hsql::Expr::OR:
            return OPERATOR_OR;
        case hsql::Expr::IN:
            return CONSTANT_ERR;
        case hsql::Expr::NOT:
            return OPERATOR_NOT;
        case hsql::Expr::UMINUS:
            return OPERATOR_UMI;
        case hsql::Expr::ISNULL:
            return CONSTANT_ERR;
        case hsql::Expr::EXISTS:
            return CONSTANT_ERR;
    }
    return CONSTANT_ERR;
}
