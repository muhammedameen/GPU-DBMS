#include "hip/hip_runtime.h"
//
// Created by gautam on 28/04/20.
//

#include <float.h>
#include "myExpr.cuh"

#define NUM_THREADS 512


myExpr *newExpr(myExprType type, long intVal) {
    auto *expr = new myExpr;
    expr->type = type;
    expr->iVal = (int)intVal;
    expr->fVal = 0.0f;
    expr->sVal[0] = 0;
    expr->childLeft = -1;
    expr->childRight = -1;
    return expr;
}

myExpr * newExpr(myExprType type, float fVal){
    auto *expr = new myExpr;
    expr->type = type;
    expr->iVal = 0;
    expr->fVal = fVal;
    expr->sVal[0] = 0;
    expr->childLeft = -1;
    expr->childRight = -1;
    return expr;
}

myExpr * newExpr(myExprType type, char *sVal){
    auto *expr = new myExpr;
    expr->type = type;
    expr->iVal = 0;
    expr->fVal = 0.0f;
    // expr->sVal = new char[strlen(sVal) + 1];
    stpcpy(expr->sVal, sVal);
    expr->childLeft = -1;
    expr->childRight = -1;
    return expr;
}

myExpr *newExpr(myExprType type){
    auto *expr = new myExpr;
    expr->type = type;
    expr->iVal = 0;
    expr->fVal = 0.0f;
    expr->sVal[0] = 0;
    expr->childLeft = -1;
    expr->childRight = -1;
    return expr;
}

void freeExpr(myExpr *expr){
    free(expr);
}

__global__ void minKernel(void *data, const int colPos, const int rowSize, const int numRows, int *min) {
    int rowsPerBlock = (numRows + NUM_THREADS - 1) / NUM_THREADS;
    unsigned int start = rowsPerBlock * threadIdx.x;
    unsigned int end = rowsPerBlock * (threadIdx.x + 1);
    int threadMin = *min;
    int *currVal;
    for (unsigned int i = start; i < end; ++i) {
        if (i >= numRows) break;
        currVal = (int *)((char *)data + i * rowSize + colPos);
        threadMin = threadMin < *currVal ? threadMin : *currVal;
    }
    atomicMin(min, threadMin);
}

__device__ float atomicMax(float* address, float val)
{
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
                          __float_as_int(::fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

__device__ float atomicMin(float* address, float val)
{
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
                          __float_as_int(::fminf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

__global__ void minKernel(void *data, const int colPos, const int rowSize, const int numRows, float *min) {
    int rowsPerBlock = (numRows + NUM_THREADS - 1) / NUM_THREADS;
    unsigned int start = rowsPerBlock * threadIdx.x;
    unsigned int end = rowsPerBlock * (threadIdx.x + 1);
    float threadMin = *min;
    float *currVal;
    for (unsigned int i = start; i < end; ++i) {
        if (i >= numRows) break;
        currVal = (float *)((char *)data + i * rowSize + colPos);
        threadMin = threadMin < *currVal ? threadMin : *currVal;
    }
    atomicMin(min, threadMin);
}

void exprToVec(hsql::Expr *expr, std::vector<myExpr> &vector, const std::vector<std::string>& colNames, Data &d) {
    switch (expr->type) {
        case hsql::kExprLiteralFloat:
            vector.push_back(*newExpr(CONSTANT_FLT, expr->fval));
            break;
        case hsql::kExprLiteralString:
            vector.push_back(*newExpr(CONSTANT_STR, expr->name));
            break;
        case hsql::kExprLiteralInt:
            vector.push_back(*newExpr(CONSTANT_INT, expr->ival));
            break;
        case hsql::kExprStar:
            printf("Why is there a `*` here?");
            break;
        case hsql::kExprPlaceholder:
            printf("What is this?");
            break;
        case hsql::kExprColumnRef:{
            int i;
            for (i = 0; i < colNames.size(); i++) {
                if (colNames[i] == expr->name) break;
            }
            vector.push_back(*newExpr(COL_NAME, (long)i));
            break;
        }
        case hsql::kExprFunctionRef: {
            // printf("%s\n", expr->name);
            int oldChunkSize = d.chunkSize;
            d.chunkSize *= 10;
            void *data = malloc(d.chunkSize * d.mdata.rowSize);
            void *data_d;
            hipMalloc(&data_d, d.chunkSize * d.mdata.rowSize);
            int rowsRead = d.read(data);
            hipMemcpy(data_d, data, rowsRead * d.mdata.rowSize, hipMemcpyHostToDevice);
            std::string colName = expr->exprList->at(0)->name;
            // printf("Col for agg function is %s:%d\n", colName.c_str(), d.mdata.colMap[colName]);
            fflush(stdout);
            int colPos = 0;
            int resType = TYPE_INT;

            for (int i = 0; i < colNames.size(); ++i) {
                if (colNames[i] == colName) {
                    resType = d.mdata.datatypes[i].type;
                    break;
                }
                colPos += d.mdata.datatypes[i].size;
            }

            if (strcmp(expr->name, "min") == 0) {
                if (resType == TYPE_INT) {
                    int min_h = INT_MAX;
                    int *min;
                    hipMalloc(&min, sizeof(int));
                    hipMemcpy(min, &min_h, sizeof(int), hipMemcpyHostToDevice);
                    while (rowsRead > 0) {
                        minKernel<<<1, NUM_THREADS>>>(data_d, colPos, d.mdata.rowSize, rowsRead, min);
                        rowsRead = d.read(data);
                        hipDeviceSynchronize();
                        hipMemcpy(data_d, data, rowsRead * d.mdata.rowSize, hipMemcpyHostToDevice);
                    }
                    hipMemcpy(&min_h, min, sizeof(int), hipMemcpyDeviceToHost);
                    hipFree(min);
                    printf("Min value is: %d\n", min_h);
                    vector.push_back(*newExpr(CONSTANT_INT, (long) min_h));
                } else if (resType == TYPE_FLOAT) {
                    float min_h = FLT_MAX;
                    float *min;
                    hipMalloc(&min, sizeof(float));
                    hipMemcpy(min, &min_h, sizeof(float), hipMemcpyHostToDevice);
                    while (rowsRead > 0) {
                        minKernel<<<1, NUM_THREADS>>>(data_d, colPos, d.mdata.rowSize, rowsRead, min);
                        rowsRead = d.read(data);
                        hipDeviceSynchronize();
                        hipMemcpy(data_d, data, rowsRead * d.mdata.rowSize, hipMemcpyHostToDevice);
                    }
                    hipMemcpy(&min_h, min, sizeof(float), hipMemcpyDeviceToHost);
                    hipFree(min);
                    vector.push_back(*newExpr(CONSTANT_FLT, min_h));
                }
            } else if (strcmp(expr->name, "max") == 0) {

            } else if (strcmp(expr->name, "sum") == 0) {

            } else if (strcmp(expr->name, "avg") == 0) {

            } else if (strcmp(expr->name, "count") == 0) {

            }
            d.chunkSize = oldChunkSize;
            d.restartRead();
            free(data);
            hipFree(data_d);
            break;
        }
        case hsql::kExprOperator: {
            myExpr *temp = newExpr(getOpType(expr->opType, expr->opChar));
            vector.push_back(*temp);
            int curr = (int)vector.size() - 1;
            vector[curr].childLeft = vector.size();
            exprToVec(expr->expr, vector, colNames, d);
            if (expr->expr2 != nullptr) {
                vector[curr].childRight = vector.size();
                exprToVec(expr->expr2, vector, colNames, d);
            }
            break;
        }
        case hsql::kExprSelect:
            printf("Not yet implemented");
            break;
    }
}

myExprType getOpType(hsql::Expr::OperatorType type, char opChar) {
    // TODO: Change Error to correct Constants
    switch (type) {
        case hsql::Expr::NONE:
            return CONSTANT_ERR;
        case hsql::Expr::BETWEEN:
            return CONSTANT_ERR;
        case hsql::Expr::CASE:
            return CONSTANT_ERR;
        case hsql::Expr::SIMPLE_OP:
            switch (opChar) {
                case '+':
                    return OPERATOR_PL;
                case '-':
                    return OPERATOR_MI;
                case '*':
                    return OPERATOR_MU;
                case '/':
                    return OPERATOR_DI;
                case '%':
                    return OPERATOR_MO;
                case '=':
                    return OPERATOR_EQ;
                case '<':
                    return OPERATOR_LT;
                case '>':
                    return OPERATOR_GT;
                default:
                    return CONSTANT_ERR;
            }
        case hsql::Expr::NOT_EQUALS:
            return OPERATOR_NE;
        case hsql::Expr::LESS_EQ:
            return OPERATOR_LE;
        case hsql::Expr::GREATER_EQ:
            return OPERATOR_GE;
        case hsql::Expr::LIKE:
            return CONSTANT_ERR;
        case hsql::Expr::NOT_LIKE:
            return CONSTANT_ERR;
        case hsql::Expr::AND:
            return OPERATOR_AND;
        case hsql::Expr::OR:
            return OPERATOR_OR;
        case hsql::Expr::IN:
            return CONSTANT_ERR;
        case hsql::Expr::NOT:
            return OPERATOR_NOT;
        case hsql::Expr::UMINUS:
            return OPERATOR_UMI;
        case hsql::Expr::ISNULL:
            return CONSTANT_ERR;
        case hsql::Expr::EXISTS:
            return CONSTANT_ERR;
    }
    return CONSTANT_ERR;
}
