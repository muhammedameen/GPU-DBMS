#include "hip/hip_runtime.h"
//
// Created by gautam on 28/04/20.
//

#include "deviceUtil.cuh"

// __device__ whereExpr *exprArr;
// __device__ ColType *types;
// __device__ int *offset;
// __device__ int whereExprSize;

void eval2(void *row, int *offset, ColType *types, whereExpr *exprArr, void *&res, int &resType);

__device__ void printRowDevice(void *row, ColType *colTypes, int numCols) {
    // int start =  0;
    // char buff[100];
    // int buffStart = 0;
    // for (int i = 0; i < numCols; i++, start += colTypes[i].size) {
    //     switch (colTypes->type) {
    //         case TYPE_INT: {
    //             int temp = *((int *) ((char *) row + start));
    //             buffStart += sprintf(buff + buffStart, "%d", temp);
    //             break;
    //         }
    //         case TYPE_FLOAT: {
    //             float temp = *((float *) ((char *) row + start));
    //             buffStart += sprintf(buff + buffStart, "%f", temp);
    //             break;
    //         }
    //         case TYPE_BOOL:
    //             break;
    //         case TYPE_VARCHAR: {
    //             char *temp = (char *) row + start;
    //             buffStart += sprintf(buff + buffStart, "%s", temp);
    //             break;
    //         }
    //         case TYPE_DATETIME:
    //             break;
    //         case TYPE_INVALID:
    //             break;
    //     }
    //     if (i != numCols - 1) {
    //         buffStart += sprintf(buff + buffStart, ", ");
    //     }
    // }
    // printf("%s\n", buff);
}

__device__ void eval2(void *row, int *offset, ColType *types, whereExpr *exprArr, void *&res, int &resType) {
    const int MAX_DEPTH = 10;
    int *exprStack = (int *) malloc(sizeof(int) * MAX_DEPTH);
    exprStack[0] = 0;        // Push Expr of 0
    bool *solved = (bool *) malloc(sizeof(bool) * 100);
    void **resArr = (void **) malloc(sizeof(void *) * 100);
    int *resTypeArr = (int *) malloc(sizeof(int) * 100);
    for (int i = 0; i < 100; i++) solved[i] = false;
    int count = 1;
    while (count > 0) {
        // pop
        int index = exprStack[count - 1];
        whereExpr *expr = &exprArr[index];
        --count;
        // check if children are solved
        // if solved evaluate current node
        // else push children
        switch (expr->type) {
            case CONSTANT_ERR:
                printf("Error");
                break;
            case CONSTANT_INT: {
                solved[index] = true;
                resTypeArr[index] = RESTYPE_INT;
                int *temp = (int *) malloc(sizeof(int));
                *temp = expr->iVal;
                resArr[index] = temp;
                break;
            }
            case OPERATOR_PL: {
                if (solved[expr->childLeft] && solved[expr->childRight]) {
                    solved[index] = true;
                    int ltype = resTypeArr[expr->childLeft];
                    int rtype = resTypeArr[expr->childRight];
                    void *lres = resArr[expr->childLeft];
                    void *rres = resArr[expr->childRight];
                    resTypeArr[index] = RESTYPE_FLT;
                    if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
                        float lhs, rhs;
                        float *temp = (float *)malloc(sizeof(float));
                        lhs = *(float *) lres;
                        rhs = *(float *) rres;
                        *temp = lhs + rhs;
                        resArr[index] = temp;
                    } else if (ltype == RESTYPE_FLT) {
                        float lhs;
                        int rhs;
                        float *temp = (float *)malloc(sizeof(float));
                        lhs = *(float *) lres;
                        rhs = *(int *) rres;
                        *temp = lhs + rhs;
                        resArr[index] = temp;
                    } else if (rtype == RESTYPE_FLT) {
                        int lhs;
                        float rhs;
                        float *temp = (float *)malloc(sizeof(float));
                        lhs = *(int *) lres;
                        rhs = *(float *) rres;
                        *temp = lhs + rhs;
                        resArr[index] = temp;
                    } else {
                        resTypeArr[index] = RESTYPE_INT;
                        int *temp = (int *)malloc(sizeof(float));
                        int lhs, rhs;
                        lhs = *(int *) lres;
                        rhs = *(int *) rres;
                        *temp = lhs + rhs;
                        resArr[index] = temp;
                    }
                } else {
                    // push this
                    // push ALL children
                    exprStack[count] = index;
                    ++count;
                    exprStack[count] = expr->childLeft;
                    ++count;
                    exprStack[count] = expr->childRight;
                    ++count;
                }
                break;
            }
            default:
                printf("Not yet implemented");
        }
    }
    // Result and restype are stored in resArr[0] and resArrType[0]
    res = resArr[0];
    resType = resTypeArr[0];
    free(resTypeArr);
    for (int i = 1; i < 100; i++) {
        if (solved[i]) {
            free(resArr[i]);
        }
    }
    free(resArr);
}

__device__ void eval(void *row, int *offset2, ColType types2[],
          whereExpr *exprArr2,
          void *&res, int &resType, int tid, bool flag) {
    if (tid == 0) {
        // offset = offset2;
        // types = types2;
        // exprArr = exprArr2;
        // whereExprSize = sizeof(whereExpr);
    }
    __syncthreads();
    if (tid == 1) {
        printf("Inside eval.\n");
        // for (int i = 0; i < 3; i++) {
        //     auto leaf = exprArr2[i];
        //     printf("TYPE: %d, ival: %d, fval: %f, sval: %s, left: %d, right: %d\n", leaf.type, leaf.iVal, leaf.fVal,
        //            leaf.sVal, leaf.childLeft, leaf.childRight);
        // }
    }
    if (flag) {
        // evalUtil(row, 0, res, resType);
        eval2(row, offset2, types2, exprArr2, res, resType);
    }
}



// __device__ void evalUtil(void *row, int currPos, void *&res, int &resType) {
//     printf("CURR POS: %d\n", currPos);
//     printf("WhereExpr size: %lu\n", sizeof(exprArr[0]));
//     whereExpr *expr = exprArr + currPos;
//     printf("Address: %ld\n", (long) &expr);
//     const auto leaf = *expr;
//     printf("TYPE inside Eval: %d, ival: %d, fval: %f, sval: %s, left: %d, right: %d\n", leaf.type, leaf.iVal, leaf.fVal,
//            leaf.sVal, leaf.childLeft, leaf.childRight);
//     switch (expr->type) {
//         case CONSTANT_ERR:
//             // printf("ERROR NOT SUPPORTED YET\n");
//             break;
//         case CONSTANT_INT:
//             // printf("INT_VAL\n");
//             // fflush(stdout);
//             res = malloc(sizeof(int));
//             resType = RESTYPE_INT;
//             memcpy(res, &expr->iVal, sizeof(int));
//             printf("val: %d\n", *((int *) res));
//             printf("val: %d\n", expr->iVal);
//             break;
//         case CONSTANT_FLT:
//             res = malloc(sizeof(float));
//             resType = RESTYPE_FLT;
//             memcpy(res, &expr->fVal, sizeof(float));
//             break;
//         case CONSTANT_STR: {
//             int len = myStrlen(expr->sVal);
//             res = malloc(sizeof(char) * len + 1);
//             resType = (int) (-len - 1);
//             memcpy(res, &expr->sVal, len + 1);
//             break;
//         }
//         case COL_NAME: {
//             int i = expr->iVal;
//             int start = offset[i];
//             int end = offset[i + 1];
//             switch (types[i].type) {
//                 case TYPE_INT:
//                     res = malloc(sizeof(int));
//                     resType = RESTYPE_INT;
//                     memcpy(res, (char *) row + start, sizeof(int));
//                     printf("Col val: %d\n", *(int *) res);
//                     break;
//                 case TYPE_FLOAT:
//                     res = malloc(sizeof(float));
//                     resType = RESTYPE_FLT;
//                     memcpy(res, (char *) row + start, sizeof(float));
//                     break;
//                 case TYPE_BOOL:
//                     printf("Not yet implemented\n");
//                     break;
//                 case TYPE_VARCHAR:
//                     res = malloc(end - start);
//                     resType = -(end - start + 1);
//                     memcpy(res, (char *) row + start, end - start);
//                     break;
//                 case TYPE_DATETIME:
//                     printf("Not yet implemented 2\n");
//                     break;
//                 case TYPE_INVALID:
//                     printf("INVALID TYPE!\n");
//                     break;
//             }
//             break;
//         }
//         case OPERATOR_PL: {
//             void *lres, *rres;
//             int ltype = 0, rtype = 0;
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childLeft, lres, ltype);
//             // size_t mem_free_0, mem_tot_0;
//             // hipMemGetInfo(&mem_free_0, &mem_tot_0);
//             printf("left: %d, right: %d\n", expr->childLeft, expr->childRight);
//             evalUtil(row, expr->childLeft, lres, ltype);
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childRight, rres, rtype);
//             evalUtil(row, expr->childRight, rres, rtype);;
//             resType = RESTYPE_FLT;
//             if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
//                 res = malloc(sizeof(float));
//                 float temp = 0;
//                 float lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = lhs + rhs;
//                 memcpy(res, &temp, sizeof(float));
//             } else if (ltype == RESTYPE_FLT) {
//                 res = malloc(sizeof(float));
//                 float temp = 0;
//                 float lhs;
//                 int rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = lhs + rhs;
//                 memcpy(res, &temp, sizeof(float));
//             } else if (rtype == RESTYPE_FLT) {
//                 res = malloc(sizeof(float));
//                 float temp = 0;
//                 int lhs;
//                 float rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = (lhs + rhs);
//                 memcpy(res, &temp, sizeof(float));
//             } else {
//                 res = malloc(sizeof(int));
//                 resType = RESTYPE_INT;
//                 int temp;
//                 int lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = (lhs + rhs);
//                 memcpy(res, &temp, sizeof(int));
//                 // printf("lhs: %d, rhs: %d", lhs, rhs);
//             }
//             // fflush(stdout);
//             free(lres);
//             free(rres);
//             break;
//         }
//         case OPERATOR_AND: {
//             void *lres, *rres;
//             int ltype = 0, rtype = 0;
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childLeft, lres, ltype);
//             evalUtil(row, expr->childLeft, lres, ltype);
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childRight, rres, rtype);
//             evalUtil(row, expr->childRight, rres, rtype);
//             res = malloc(sizeof(int));
//             resType = RESTYPE_INT;
//             int temp = 0;
//             if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
//                 float lhs, rhs;
//                 lhs = *(float *) lres;
//                 rhs = *(float *) rres;
//                 // memcpy(&lhs, lres, sizeof(float));
//                 // memcpy(&rhs, rres, sizeof(float));
//                 temp = lhs && rhs;
//             } else if (ltype == RESTYPE_FLT) {
//                 float lhs;
//                 int rhs;
//                 lhs = *(float *) lres;
//                 rhs = *(int *) rres;
//                 // memcpy(&lhs, lres, sizeof(float));
//                 // memcpy(&rhs, rres, sizeof(int));
//                 temp = lhs && rhs;
//             } else if (rtype == RESTYPE_FLT) {
//                 int lhs;
//                 float rhs;
//                 lhs = *(int *) lres;
//                 rhs = *(float *) rres;
//                 // memcpy(&lhs, lres, sizeof(int));
//                 // memcpy(&rhs, rres, sizeof(float));
//                 temp = lhs && rhs;
//             } else {
//                 int lhs, rhs;
//                 lhs = *(int *) lres;
//                 rhs = *(int *) rres;
//                 // memcpy(&lhs, lres, sizeof(int));
//                 // memcpy(&rhs, rres, sizeof(int));
//                 temp = lhs && rhs;
//             }
//             memcpy(res, &temp, sizeof(int));
//             // printf("INSIDE AND %d\n", temp);
//             // fflush(stdout);
//             free(lres);
//             free(rres);
//             break;
//         }
//         case OPERATOR_OR: {
//             void *lres, *rres;
//             int ltype = 0, rtype = 0;
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childLeft, lres, ltype);
//             evalUtil(row, expr->childLeft, lres, ltype);
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childRight, rres, rtype);
//             evalUtil(row, expr->childRight, rres, rtype);
//             res = malloc(sizeof(int));
//             resType = RESTYPE_INT;
//             int temp = 0;
//             if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
//                 float lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = lhs || rhs;
//             } else if (ltype == RESTYPE_FLT) {
//                 float lhs;
//                 int rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = lhs || rhs;
//             } else if (rtype == RESTYPE_FLT) {
//                 int lhs;
//                 float rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = lhs || rhs;
//             } else {
//                 int lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = lhs || rhs;
//             }
//             memcpy(res, &temp, sizeof(int));
//             // printf("INSIDE AND %d\n", temp);
//             // fflush(stdout);
//             free(lres);
//             free(rres);
//             break;
//         }
//         case OPERATOR_NOT: {
//             void *lres;
//             int ltype = 0;
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childLeft, lres, ltype);
//             evalUtil(row, expr->childLeft, lres, ltype);
//             res = malloc(sizeof(int));
//             resType = RESTYPE_INT;
//             int temp = 0;
//             if (ltype == RESTYPE_FLT) {
//                 float lhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 temp = !lhs;
//             } else {
//                 int lhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 temp = !lhs;
//                 // printf("lhs: %d, rhs: %d", lhs, rhs);
//             }
//             memcpy(res, &temp, sizeof(int));
//             // printf("INSIDE = %d\n", temp);
//             // fflush(stdout);
//             free(lres);
//             break;
//         }
//         case OPERATOR_EQ: {
//             void *lres, *rres;
//             int ltype = 0, rtype = 0;
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childLeft, lres, ltype);
//             evalUtil(row, expr->childLeft, lres, ltype);
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childRight, rres, rtype);
//             evalUtil(row, expr->childRight, rres, rtype);
//             res = malloc(sizeof(int));
//             resType = RESTYPE_INT;
//             int temp = 0;
//             if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
//                 float lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = lhs == rhs;
//             } else if (ltype == RESTYPE_FLT) {
//                 float lhs;
//                 int rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = lhs == rhs;
//             } else if (rtype == RESTYPE_FLT) {
//                 int lhs;
//                 float rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = lhs == rhs;
//             } else {
//                 int lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = lhs == rhs;
//                 // printf("lhs: %d, rhs: %d", lhs, rhs);
//             }
//             memcpy(res, &temp, sizeof(int));
//             // printf("INSIDE = %d\n", temp);
//             // fflush(stdout);
//             free(lres);
//             free(rres);
//             break;
//         }
//         case OPERATOR_NE: {
//             void *lres, *rres;
//             int ltype = 0, rtype = 0;
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childLeft, lres, ltype);
//             evalUtil(row, expr->childLeft, lres, ltype);
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childRight, rres, rtype);
//             evalUtil(row, expr->childRight, rres, rtype);
//             res = malloc(sizeof(int));
//             resType = RESTYPE_INT;
//             int temp = 0;
//             if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
//                 float lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = (lhs != rhs);
//             } else if (ltype == RESTYPE_FLT) {
//                 float lhs;
//                 int rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = (lhs != rhs);
//             } else if (rtype == RESTYPE_FLT) {
//                 int lhs;
//                 float rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = (lhs != rhs);
//             } else {
//                 int lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = (lhs != rhs);
//                 // printf("lhs: %d, rhs: %d", lhs, rhs);
//             }
//             memcpy(res, &temp, sizeof(int));
//             // printf("INSIDE = %d\n", temp);
//             // fflush(stdout);
//             free(lres);
//             free(rres);
//             break;
//         }
//         case OPERATOR_GE: {
//             void *lres, *rres;
//             int ltype = 0, rtype = 0;
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childLeft, lres, ltype);
//             evalUtil(row, expr->childLeft, lres, ltype);
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childRight, rres, rtype);
//             evalUtil(row, expr->childRight, rres, rtype);;
//             res = malloc(sizeof(int));
//             resType = RESTYPE_INT;
//             int temp = 0;
//             if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
//                 float lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = (lhs >= rhs);
//             } else if (ltype == RESTYPE_FLT) {
//                 float lhs;
//                 int rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = (lhs >= rhs);
//             } else if (rtype == RESTYPE_FLT) {
//                 int lhs;
//                 float rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = (lhs >= rhs);
//             } else {
//                 int lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = (lhs >= rhs);
//                 // printf("lhs: %d, rhs: %d", lhs, rhs);
//             }
//             memcpy(res, &temp, sizeof(int));
//             // printf("INSIDE = %d\n", temp);
//             // fflush(stdout);
//             free(lres);
//             free(rres);
//             break;
//         }
//         case OPERATOR_LE: {
//             void *lres, *rres;
//             int ltype = 0, rtype = 0;
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childLeft, lres, ltype);
//             evalUtil(row, expr->childLeft, lres, ltype);
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childRight, rres, rtype);
//             evalUtil(row, expr->childRight, rres, rtype);;
//             res = malloc(sizeof(int));
//             resType = RESTYPE_INT;
//             int temp = 0;
//             if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
//                 float lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = (lhs <= rhs);
//             } else if (ltype == RESTYPE_FLT) {
//                 float lhs;
//                 int rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = (lhs <= rhs);
//             } else if (rtype == RESTYPE_FLT) {
//                 int lhs;
//                 float rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = (lhs <= rhs);
//             } else {
//                 int lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = (lhs <= rhs);
//                 // printf("lhs: %d, rhs: %d", lhs, rhs);
//             }
//             memcpy(res, &temp, sizeof(int));
//             // printf("INSIDE = %d\n", temp);
//             // fflush(stdout);
//             free(lres);
//             free(rres);
//             break;
//         }
//         case OPERATOR_GT: {
//             void *lres, *rres;
//             int ltype = 0, rtype = 0;
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childLeft, lres, ltype);
//             evalUtil(row, expr->childLeft, lres, ltype);
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childRight, rres, rtype);
//             evalUtil(row, expr->childRight, rres, rtype);;
//             res = malloc(sizeof(int));
//             resType = RESTYPE_INT;
//             int temp = 0;
//             if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
//                 float lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = (lhs > rhs);
//             } else if (ltype == RESTYPE_FLT) {
//                 float lhs;
//                 int rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = (lhs > rhs);
//             } else if (rtype == RESTYPE_FLT) {
//                 int lhs;
//                 float rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = (lhs > rhs);
//             } else {
//                 int lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = (lhs > rhs);
//                 // printf("lhs: %d, rhs: %d", lhs, rhs);
//             }
//             memcpy(res, &temp, sizeof(int));
//             // printf("INSIDE = %d\n", temp);
//             // fflush(stdout);
//             free(lres);
//             free(rres);
//             break;
//         }
//         case OPERATOR_LT: {
//             void *lres, *rres;
//             int ltype = 0, rtype = 0;
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childLeft, lres, ltype);
//             evalUtil(row, expr->childLeft, lres, ltype);
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childRight, rres, rtype);
//             evalUtil(row, expr->childRight, rres, rtype);;
//             res = malloc(sizeof(int));
//             resType = RESTYPE_INT;
//             int temp = 0;
//             if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
//                 float lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = (lhs < rhs);
//             } else if (ltype == RESTYPE_FLT) {
//                 float lhs;
//                 int rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = (lhs < rhs);
//             } else if (rtype == RESTYPE_FLT) {
//                 int lhs;
//                 float rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = (lhs < rhs);
//             } else {
//                 int lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = (lhs < rhs);
//                 // printf("lhs: %d, rhs: %d", lhs, rhs);
//             }
//             memcpy(res, &temp, sizeof(int));
//             // printf("INSIDE = %d\n", temp);
//             // fflush(stdout);
//             free(lres);
//             free(rres);
//             break;
//         }
//         case OPERATOR_MI: {
//             void *lres, *rres;
//             int ltype = 0, rtype = 0;
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childLeft, lres, ltype);
//             evalUtil(row, expr->childLeft, lres, ltype);
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childRight, rres, rtype);
//             evalUtil(row, expr->childRight, rres, rtype);
//             res = malloc(sizeof(int));
//             resType = RESTYPE_FLT;
//             if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
//                 float temp = 0;
//                 float lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = lhs - rhs;
//                 memcpy(res, &temp, sizeof(float));
//             } else if (ltype == RESTYPE_FLT) {
//                 float temp = 0;
//                 float lhs;
//                 int rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = lhs - rhs;
//                 memcpy(res, &temp, sizeof(float));
//             } else if (rtype == RESTYPE_FLT) {
//                 float temp = 0;
//                 int lhs;
//                 float rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = (lhs - rhs);
//                 memcpy(res, &temp, sizeof(float));
//             } else {
//                 resType = RESTYPE_INT;
//                 int temp;
//                 int lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = (lhs - rhs);
//                 memcpy(res, &temp, sizeof(int));
//                 // printf("lhs: %d, rhs: %d", lhs, rhs);
//             }
//             // fflush(stdout);
//             free(lres);
//             free(rres);
//             break;
//         }
//         case OPERATOR_MU: {
//             void *lres, *rres;
//             int ltype = 0, rtype = 0;
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childLeft, lres, ltype);
//             evalUtil(row, expr->childLeft, lres, ltype);
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childRight, rres, rtype);
//             evalUtil(row, expr->childRight, rres, rtype);
//             res = malloc(sizeof(int));
//             resType = RESTYPE_FLT;
//             if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
//                 float temp = 0;
//                 float lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = lhs * rhs;
//                 memcpy(res, &temp, sizeof(float));
//             } else if (ltype == RESTYPE_FLT) {
//                 float temp = 0;
//                 float lhs;
//                 int rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = lhs * rhs;
//                 memcpy(res, &temp, sizeof(float));
//             } else if (rtype == RESTYPE_FLT) {
//                 float temp = 0;
//                 int lhs;
//                 float rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = (lhs * rhs);
//                 memcpy(res, &temp, sizeof(float));
//                 // printf("Value : !!%f!!", temp);
//             } else {
//                 resType = RESTYPE_INT;
//                 int temp;
//                 int lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(int));
//                 printf("lhs: %d, rhs: %d", lhs, rhs);
//                 temp = (lhs * rhs);
//                 memcpy(res, &temp, sizeof(int));
//             }
//             // fflush(stdout);
//             free(lres);
//             free(rres);
//             break;
//         }
//         case OPERATOR_DI: {
//             void *lres, *rres;
//             int ltype = 0, rtype = 0;
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childLeft, lres, ltype);
//             evalUtil(row, expr->childLeft, lres, ltype);
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childRight, rres, rtype);
//             evalUtil(row, expr->childRight, rres, rtype);
//             res = malloc(sizeof(int));
//             resType = RESTYPE_FLT;
//             if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
//                 float temp = 0;
//                 float lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = lhs / rhs;
//                 memcpy(res, &temp, sizeof(float));
//             } else if (ltype == RESTYPE_FLT) {
//                 float temp = 0;
//                 float lhs;
//                 int rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = lhs / rhs;
//                 memcpy(res, &temp, sizeof(float));
//             } else if (rtype == RESTYPE_FLT) {
//                 float temp = 0;
//                 int lhs;
//                 float rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = (lhs / rhs);
//                 memcpy(res, &temp, sizeof(float));
//             } else {
//                 resType = RESTYPE_INT;
//                 int temp;
//                 int lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = (lhs / rhs);
//                 memcpy(res, &temp, sizeof(int));
//                 // printf("lhs: %d, rhs: %d", lhs, rhs);
//             }
//             // fflush(stdout);
//             free(lres);
//             free(rres);
//             break;
//         }
//         case OPERATOR_MO: {
//             void *lres, *rres;
//             int ltype = 0, rtype = 0;
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childLeft, lres, ltype);
//             evalUtil(row, expr->childLeft, lres, ltype);
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childRight, rres, rtype);
//             evalUtil(row, expr->childRight, rres, rtype);
//             res = malloc(sizeof(int));
//             resType = RESTYPE_INT;
//             int temp = 0;
//             if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
//                 float lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = (int) lhs % (int) rhs;
//             } else if (ltype == RESTYPE_FLT) {
//                 float lhs;
//                 int rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = (int) lhs % rhs;
//             } else if (rtype == RESTYPE_FLT) {
//                 int lhs;
//                 float rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = lhs % (int) rhs;
//             } else {
//                 int lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = lhs % rhs;
//                 // printf("lhs: %d, rhs: %d", lhs, rhs);
//             }
//             memcpy(res, &temp, sizeof(int));
//             // fflush(stdout);
//             free(lres);
//             free(rres);
//             break;
//         }
//         case OPERATOR_UMI: {
//             void *lres;
//             int ltype = 0;
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childLeft, lres, ltype);
//             evalUtil(row, expr->childLeft, lres, ltype);
//             res = malloc(sizeof(int));
//             resType = RESTYPE_FLT;
//             if (ltype == RESTYPE_FLT) {
//                 float temp = 0;
//                 float lhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 temp = -lhs;
//                 memcpy(res, &temp, sizeof(int));
//             } else {
//                 resType = RESTYPE_INT;
//                 int temp;
//                 int lhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 temp = -lhs;
//                 memcpy(res, &temp, sizeof(int));
//                 // printf("lhs: %d, rhs: %d", lhs, rhs);
//             }
//             // fflush(stdout);
//             free(lres);
//             break;
//         }
//     }
// }

__device__ int myStrncmp(const char *str_a, const char *str_b, unsigned len) {
    int match = 0;
    unsigned i = 0;
    unsigned done = 0;
    while ((i < len) && (match == 0) && !done) {
        if ((str_a[i] == 0) || (str_b[i] == 0)) done = 1;
        else if (str_a[i] != str_b[i]) {
            match = i + 1;
            if ((int) str_a[i] - (int) str_b[i] < 0) match = 0 - (i + 1);
        }
        i++;
    }
    return match;
}

__device__ int myStrlen(const char *str) {
    int length = 0;
    while (str[length++] != 0);
    return length;
}