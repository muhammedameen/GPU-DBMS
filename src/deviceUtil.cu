#include "hip/hip_runtime.h"
//
// Created by gautam on 28/04/20.
//

#include "deviceUtil.cuh"

__device__ int printStar(void *row, ColType *colTypes, int numCols, char *buff) {
    int start =  0;
    // char buff[100];
    int buffStart = 0;
    // float flt;
    // memcpy(&flt, (char *) row + 12, sizeof(float));
    // float * fltptr = (float *)((char *)row + 12);
    // printf("Float: %f, %f\n", *fltptr, flt);
    for (int i = 0; i < numCols; start += colTypes[i].size, i++) {
        // printf("i: %d, start: %d\n", i, start);
        switch (colTypes[i].type) {
            case TYPE_INT: {
                // int *temp = ((int *) ((char *) row + start));
                int temp;
                memcpy(&temp, (char *)row + start, sizeof(int));
                if (!isNull(&temp)) {
                    buffStart += appendInt(buff + buffStart, temp);
                } else {
                    buffStart += appendStr(buff + buffStart, "NULL");
                }
                break;
            }
            case TYPE_FLOAT: {
                float temp;
                memcpy(&temp, (char *) row + start, sizeof(float));
                // printf("FLOAT VAL: %f, %f\n", temp, *(float *)((char *) row + start));
                // printf("FLOAT USING MEMCPY: %f\n", temp2);
                // float *temp = ((float *) ((char *) row + start));

                if (!isNull(&temp)) {
                    buffStart += appendFlt(buff + buffStart, temp);
                    // printf("Float val: %f\n", temp);
                } else {
                    buffStart += appendStr(buff + buffStart, "NULL");
                }
                break;
            }
            case TYPE_VARCHAR: {
                char *temp = (char *) row + start;

                if (!isNull(temp)) {
                    buffStart += appendStr(buff + buffStart, temp);
                } else {
                    buffStart += appendStr(buff + buffStart, "NULL");
                }
                break;
            }
            default:
                printf("Not yet implemented");
        }
        if (i != numCols - 1) {
            buffStart += appendStr(buff + buffStart, ", ");
        }
    }
    buff[buffStart] = 0;
    return buffStart;
    // printf("%s\n", buff);
}

__device__ void printRowDevice(void *row, ColType *colTypes, int numCols, const int *dispCols, int numDispCols, const int *offsets) {
    char *buff = (char *)malloc(sizeof(char) * 250);
    int buffStart = 0;
    for (int i = 0; i < numDispCols; ++i) {
        if (dispCols[i] == -1) {
            buffStart += printStar(row, colTypes, numCols, buff + buffStart);
        } else if (dispCols[i] == -2) {
            printf("Not implemented");
            free(buff);
            return;
        } else {
            int colId = dispCols[i];
            switch (colTypes[colId].type) {
                case TYPE_INT: {
                    // int *temp = ((int *) ((char *) row + offsets[colId]));
                    int temp;
                    memcpy(&temp, (char *)row + offsets[colId], sizeof(int));
                    if (!isNull(&temp)) {
                        buffStart += appendInt(buff + buffStart, temp);
                    } else {
                        buffStart += appendStr(buff + buffStart, "NULL");
                    }
                    break;
                }
                case TYPE_FLOAT: {
                    // float *temp = ((float *) ((char *) row + offsets[colId]));
                    float temp;
                    memcpy(&temp, (char *)row + offsets[colId], sizeof(float));
                    if (!isNull(&temp)) {
                        buffStart += appendFlt(buff + buffStart, temp);
                    } else {
                        buffStart += appendStr(buff + buffStart, "NULL");
                    }
                    break;
                }
                case TYPE_VARCHAR: {
                    char *temp = (char *) row + offsets[colId];
                    if (!isNull(temp)) {
                        buffStart += appendStr(buff + buffStart, temp);
                    } else {
                        buffStart += appendStr(buff + buffStart, "NULL");
                    }
                    break;
                }
                default:
                    printf("Not yet implemented");
            }
        }
        if (i != numDispCols - 1) {
            buffStart += appendStr(buff + buffStart, ", ");
        }
    }
    buff[buffStart] = 0;
    printf("%s\n", buff);
    free(buff);
}

__device__ void eval(void *row, int *offset, ColType *types, myExpr *exprArr, void *&res, int &resType) {
    const int MAX_DEPTH = 50;
    int *exprStack = (int *) malloc(sizeof(int) * MAX_DEPTH);
    exprStack[0] = 0;        // Push Expr of 0
    bool *solved = (bool *) malloc(sizeof(bool) * 100);
    void **resArr = (void **) malloc(sizeof(void *) * 100);
    int *resTypeArr = (int *) malloc(sizeof(int) * 100);
    bool *freeAble = (bool *) malloc(sizeof(bool) * 100);
    for (int i = 0; i < 100; i++) {solved[i] = false; freeAble[i] = true;}
    int count = 1;
    while (count > 0) {
        // pop
        int index = exprStack[count - 1];
        myExpr *expr = &exprArr[index];
        --count;
        // check if children are solved; if solved evaluate current node; else push children
        switch (expr->type) {
            case CONSTANT_ERR:
                printf("Error");
                break;
            case CONSTANT_INT: {
                solved[index] = true;
                resTypeArr[index] = RESTYPE_INT;
                int *temp = (int *) malloc(sizeof(int));
                *temp = expr->iVal;
                resArr[index] = temp;
                break;
            }
            case CONSTANT_FLT: {
                solved[index] = true;
                resTypeArr[index] = RESTYPE_FLT;
                float *temp = (float *) malloc(sizeof(float));
                *temp = expr->fVal;
                resArr[index] = temp;
                break;
            }
            case CONSTANT_STR:{
                solved[index] = true;
                resTypeArr[index] = -myStrlen(expr->sVal) - 1;
                resArr[index] = expr->sVal;
                freeAble[index] = false;
                break;
            }
            case COL_NAME: {
                int colId = expr->iVal;
                int start = offset[colId];
                solved[index] = true;
                resArr[index] = (char *) row + start;
                freeAble[index] = false;
                switch (types[colId].type) {
                    case TYPE_INT: {
                        resTypeArr[index] = RESTYPE_INT;
                        break;
                    }
                    case TYPE_FLOAT: {
                        resTypeArr[index] = RESTYPE_FLT;
                        break;
                    }
                    case TYPE_VARCHAR: {
                        resTypeArr[index] = types[colId].size;
                        break;
                    }
                    default:
                        printf("Not implemented\n");
                }
            }
            case OPERATOR_PL: {
                if (solved[expr->childLeft] && solved[expr->childRight]) {
                    solved[index] = true;
                    int ltype = resTypeArr[expr->childLeft];
                    int rtype = resTypeArr[expr->childRight];
                    void *lres = resArr[expr->childLeft];
                    void *rres = resArr[expr->childRight];
                    resTypeArr[index] = RESTYPE_FLT;
                    if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
                        float lhs, rhs;
                        float *temp = (float *) malloc(sizeof(float));
                        lhs = *(float *) lres;
                        rhs = *(float *) rres;
                        *temp = lhs + rhs;
                        resArr[index] = temp;
                    } else if (ltype == RESTYPE_FLT) {
                        float lhs;
                        int rhs;
                        float *temp = (float *) malloc(sizeof(float));
                        lhs = *(float *) lres;
                        rhs = *(int *) rres;
                        *temp = lhs + rhs;
                        resArr[index] = temp;
                    } else if (rtype == RESTYPE_FLT) {
                        int lhs;
                        float rhs;
                        float *temp = (float *) malloc(sizeof(float));
                        lhs = *(int *) lres;
                        rhs = *(float *) rres;
                        *temp = lhs + rhs;
                        resArr[index] = temp;
                    } else {
                        resTypeArr[index] = RESTYPE_INT;
                        int *temp = (int *) malloc(sizeof(float));
                        int lhs, rhs;
                        lhs = *(int *) lres;
                        rhs = *(int *) rres;
                        *temp = lhs + rhs;
                        resArr[index] = temp;
                    }
                }
                break;
            }
            case OPERATOR_EQ: {
                if (solved[expr->childLeft] && solved[expr->childRight]) {
                    solved[index] = true;
                    int ltype = resTypeArr[expr->childLeft];
                    int rtype = resTypeArr[expr->childRight];
                    void *lres = resArr[expr->childLeft];
                    void *rres = resArr[expr->childRight];
                    resTypeArr[index] = RESTYPE_INT;
                    int *temp = (int *) malloc(sizeof(int));
                    if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
                        float lhs, rhs;
                        lhs = *(float *) lres;
                        rhs = *(float *) rres;
                        *temp = lhs == rhs;
                    } else if (ltype == RESTYPE_FLT && rtype == RESTYPE_INT) {
                        float lhs;
                        int rhs;
                        lhs = *(float *) lres;
                        rhs = *(int *) rres;
                        *temp = lhs == rhs;
                    } else if (rtype == RESTYPE_FLT && ltype == RESTYPE_INT) {
                        int lhs;
                        float rhs;
                        lhs = *(int *) lres;
                        rhs = *(float *) rres;
                        *temp = lhs == rhs;
                    } else if (ltype == RESTYPE_INT && rtype == RESTYPE_INT){
                        int lhs, rhs;
                        lhs = *(int *) lres;
                        rhs = *(int *) rres;
                        *temp = lhs == rhs;
                    } else {
                        *temp = myStrncmp((const char *) lres, (const char *) rres, min(-ltype, -rtype)) == 0;
                    }
                    resArr[index] = temp;
                }
                break;
            }
            case OPERATOR_MI: {
                if (solved[expr->childLeft] && solved[expr->childRight]) {
                    solved[index] = true;
                    int ltype = resTypeArr[expr->childLeft];
                    int rtype = resTypeArr[expr->childRight];
                    void *lres = resArr[expr->childLeft];
                    void *rres = resArr[expr->childRight];
                    resTypeArr[index] = RESTYPE_FLT;
                    if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
                        float lhs, rhs;
                        float *temp = (float *)malloc(sizeof(float));
                        lhs = *(float *) lres;
                        rhs = *(float *) rres;
                        *temp = lhs - rhs;
                        resArr[index] = temp;
                    } else if (ltype == RESTYPE_FLT) {
                        float lhs;
                        int rhs;
                        float *temp = (float *)malloc(sizeof(float));
                        lhs = *(float *) lres;
                        rhs = *(int *) rres;
                        *temp = lhs - rhs;
                        resArr[index] = temp;
                    } else if (rtype == RESTYPE_FLT) {
                        int lhs;
                        float rhs;
                        float *temp = (float *)malloc(sizeof(float));
                        lhs = *(int *) lres;
                        rhs = *(float *) rres;
                        *temp = lhs - rhs;
                        resArr[index] = temp;
                    } else {
                        resTypeArr[index] = RESTYPE_INT;
                        int *temp = (int *)malloc(sizeof(int));
                        int lhs, rhs;
                        lhs = *(int *) lres;
                        rhs = *(int *) rres;
                        *temp = lhs - rhs;
                        resArr[index] = temp;
                    }
                }
                break;
            }
            case OPERATOR_MU: {
                if (solved[expr->childLeft] && solved[expr->childRight]) {
                    solved[index] = true;
                    int ltype = resTypeArr[expr->childLeft];
                    int rtype = resTypeArr[expr->childRight];
                    void *lres = resArr[expr->childLeft];
                    void *rres = resArr[expr->childRight];
                    resTypeArr[index] = RESTYPE_FLT;
                    if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
                        float lhs, rhs;
                        float *temp = (float *)malloc(sizeof(float));
                        lhs = *(float *) lres;
                        rhs = *(float *) rres;
                        *temp = lhs * rhs;
                        resArr[index] = temp;
                    } else if (ltype == RESTYPE_FLT) {
                        float lhs;
                        int rhs;
                        float *temp = (float *)malloc(sizeof(float));
                        lhs = *(float *) lres;
                        rhs = *(int *) rres;
                        *temp = lhs * rhs;
                        resArr[index] = temp;
                    } else if (rtype == RESTYPE_FLT) {
                        int lhs;
                        float rhs;
                        float *temp = (float *)malloc(sizeof(float));
                        lhs = *(int *) lres;
                        rhs = *(float *) rres;
                        *temp = lhs * rhs;
                        resArr[index] = temp;
                    } else {
                        resTypeArr[index] = RESTYPE_INT;
                        int *temp = (int *)malloc(sizeof(int));
                        int lhs, rhs;
                        lhs = *(int *) lres;
                        rhs = *(int *) rres;
                        *temp = lhs * rhs;
                        resArr[index] = temp;
                    }
                }
                break;
            }
            case OPERATOR_DI: {
                if (solved[expr->childLeft] && solved[expr->childRight]) {
                    solved[index] = true;
                    int ltype = resTypeArr[expr->childLeft];
                    int rtype = resTypeArr[expr->childRight];
                    void *lres = resArr[expr->childLeft];
                    void *rres = resArr[expr->childRight];
                    resTypeArr[index] = RESTYPE_FLT;
                    if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
                        float lhs, rhs;
                        float *temp = (float *)malloc(sizeof(float));
                        lhs = *(float *) lres;
                        rhs = *(float *) rres;
                        *temp = lhs / rhs;
                        resArr[index] = temp;
                    } else if (ltype == RESTYPE_FLT) {
                        float lhs;
                        int rhs;
                        float *temp = (float *)malloc(sizeof(float));
                        lhs = *(float *) lres;
                        rhs = *(int *) rres;
                        *temp = lhs / rhs;
                        resArr[index] = temp;
                    } else if (rtype == RESTYPE_FLT) {
                        int lhs;
                        float rhs;
                        float *temp = (float *)malloc(sizeof(float));
                        lhs = *(int *) lres;
                        rhs = *(float *) rres;
                        *temp = lhs / rhs;
                        resArr[index] = temp;
                    } else {
                        resTypeArr[index] = RESTYPE_INT;
                        int *temp = (int *)malloc(sizeof(int));
                        int lhs, rhs;
                        lhs = *(int *) lres;
                        rhs = *(int *) rres;
                        *temp = lhs / rhs;
                        resArr[index] = temp;
                    }
                }
                break;
            }
            case OPERATOR_GT: {
                if (solved[expr->childLeft] && solved[expr->childRight]) {
                    solved[index] = true;
                    int ltype = resTypeArr[expr->childLeft];
                    int rtype = resTypeArr[expr->childRight];
                    void *lres = resArr[expr->childLeft];
                    void *rres = resArr[expr->childRight];
                    resTypeArr[index] = RESTYPE_INT;
                    int *temp = (int *)malloc(sizeof(int));
                    if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
                        float lhs, rhs;
                        lhs = *(float *) lres;
                        rhs = *(float *) rres;
                        *temp = lhs > rhs;
                        resArr[index] = temp;
                    } else if (ltype == RESTYPE_FLT) {
                        float lhs;
                        int rhs;
                        lhs = *(float *) lres;
                        rhs = *(int *) rres;
                        *temp = lhs > rhs;
                        resArr[index] = temp;
                    } else if (rtype == RESTYPE_FLT) {
                        int lhs;
                        float rhs;
                        lhs = *(int *) lres;
                        rhs = *(float *) rres;
                        *temp = lhs > rhs;
                        resArr[index] = temp;
                    } else {
                        resTypeArr[index] = RESTYPE_INT;
                        int lhs, rhs;
                        lhs = *(int *) lres;
                        rhs = *(int *) rres;
                        *temp = lhs > rhs;
                        resArr[index] = temp;
                    }
                }
                break;
            }
            case OPERATOR_GE: {
                if (solved[expr->childLeft] && solved[expr->childRight]) {
                    solved[index] = true;
                    int ltype = resTypeArr[expr->childLeft];
                    int rtype = resTypeArr[expr->childRight];
                    void *lres = resArr[expr->childLeft];
                    void *rres = resArr[expr->childRight];
                    resTypeArr[index] = RESTYPE_INT;
                    int *temp = (int *)malloc(sizeof(int));
                    if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
                        float lhs, rhs;
                        lhs = *(float *) lres;
                        rhs = *(float *) rres;
                        *temp = lhs >= rhs;
                        resArr[index] = temp;
                    } else if (ltype == RESTYPE_FLT) {
                        float lhs;
                        int rhs;
                        lhs = *(float *) lres;
                        rhs = *(int *) rres;
                        *temp = lhs >= rhs;
                        resArr[index] = temp;
                    } else if (rtype == RESTYPE_FLT) {
                        int lhs;
                        float rhs;
                        lhs = *(int *) lres;
                        rhs = *(float *) rres;
                        *temp = lhs >= rhs;
                        resArr[index] = temp;
                    } else {
                        resTypeArr[index] = RESTYPE_INT;
                        int lhs, rhs;
                        lhs = *(int *) lres;
                        rhs = *(int *) rres;
                        *temp = lhs >= rhs;
                        resArr[index] = temp;
                    }
                }
                break;
            }
            case OPERATOR_LT: {
                if (solved[expr->childLeft] && solved[expr->childRight]) {
                    solved[index] = true;
                    int ltype = resTypeArr[expr->childLeft];
                    int rtype = resTypeArr[expr->childRight];
                    void *lres = resArr[expr->childLeft];
                    void *rres = resArr[expr->childRight];
                    resTypeArr[index] = RESTYPE_INT;
                    int *temp = (int *)malloc(sizeof(int));
                    if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
                        float lhs, rhs;
                        lhs = *(float *) lres;
                        rhs = *(float *) rres;
                        *temp = lhs < rhs;
                        resArr[index] = temp;
                    } else if (ltype == RESTYPE_FLT) {
                        float lhs;
                        int rhs;
                        lhs = *(float *) lres;
                        rhs = *(int *) rres;
                        *temp = lhs < rhs;
                        resArr[index] = temp;
                    } else if (rtype == RESTYPE_FLT) {
                        int lhs;
                        float rhs;
                        lhs = *(int *) lres;
                        rhs = *(float *) rres;
                        *temp = lhs < rhs;
                        resArr[index] = temp;
                    } else {
                        resTypeArr[index] = RESTYPE_INT;
                        int lhs, rhs;
                        lhs = *(int *) lres;
                        rhs = *(int *) rres;
                        *temp = lhs < rhs;
                        resArr[index] = temp;
                    }
                }
                break;
            }
            case OPERATOR_LE: {
                if (solved[expr->childLeft] && solved[expr->childRight]) {
                    solved[index] = true;
                    int ltype = resTypeArr[expr->childLeft];
                    int rtype = resTypeArr[expr->childRight];
                    void *lres = resArr[expr->childLeft];
                    void *rres = resArr[expr->childRight];
                    resTypeArr[index] = RESTYPE_INT;
                    int *temp = (int *)malloc(sizeof(int));
                    if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
                        float lhs, rhs;
                        lhs = *(float *) lres;
                        rhs = *(float *) rres;
                        *temp = lhs <= rhs;
                        resArr[index] = temp;
                    } else if (ltype == RESTYPE_FLT) {
                        float lhs;
                        int rhs;
                        lhs = *(float *) lres;
                        rhs = *(int *) rres;
                        *temp = lhs <= rhs;
                        resArr[index] = temp;
                    } else if (rtype == RESTYPE_FLT) {
                        int lhs;
                        float rhs;
                        lhs = *(int *) lres;
                        rhs = *(float *) rres;
                        *temp = lhs <= rhs;
                        resArr[index] = temp;
                    } else {
                        resTypeArr[index] = RESTYPE_INT;
                        int lhs, rhs;
                        lhs = *(int *) lres;
                        rhs = *(int *) rres;
                        *temp = lhs <= rhs;
                        resArr[index] = temp;
                    }
                }
                break;
            }
            case OPERATOR_MO: {
                if (solved[expr->childLeft] && solved[expr->childRight]) {
                    solved[index] = true;
                    // int ltype = resTypeArr[expr->childLeft];
                    // int rtype = resTypeArr[expr->childRight];
                    void *lres = resArr[expr->childLeft];
                    void *rres = resArr[expr->childRight];
                    resTypeArr[index] = RESTYPE_INT;
                    int *temp = (int *)malloc(sizeof(int));
                    int lhs, rhs;
                    lhs = *(int *) lres;
                    rhs = *(int *) rres;
                    *temp = lhs % rhs;
                    resArr[index] = temp;
                }
                break;
            }
            default:
                printf("Not yet implemented");
        }
        if(!solved[index]){
            // push this and ALL children
            exprStack[count] = index;
            ++count;
            if (expr->childLeft != -1) {
                exprStack[count] = expr->childLeft;
                ++count;
            }
            if (expr->childRight != -1) {
                exprStack[count] = expr->childRight;
                ++count;
            }
            continue;
        }
    }
    // Result and restype are stored in resArr[0] and resArrType[0]
    res = resArr[0];
    resType = resTypeArr[0];
    free(resTypeArr);
    for (int i = 1; i < 100; i++) {
        if (solved[i] && freeAble[i]) {
            free(resArr[i]);
        }
    }
    free(resArr);
}

__device__ int myStrncmp(const char *str_a, const char *str_b, unsigned len) {
    int match = 0;
    unsigned i = 0;
    unsigned done = 0;
    while ((i < len) && (match == 0) && !done) {
        if ((str_a[i] == 0) || (str_b[i] == 0)) done = 1;
        else if (str_a[i] != str_b[i]) {
            match = i + 1;
            if ((int) str_a[i] - (int) str_b[i] < 0) match = 0 - (i + 1);
        }
        i++;
    }
    return match;
}

__device__ int myStrlen(const char *str) {
    int length = 0;
    while (str[length++] != 0);
    return length;
}

__device__ void reverse(char* str, int len)
{
    int i = 0, j = len - 1, temp;
    while (i < j) {
        temp = str[i];
        str[i] = str[j];
        str[j] = temp;
        i++;
        j--;
    }
}

// Implementation of itoa()
__device__ char* itoa(int num, char* str)
{
    int base = 10;
    int i = 0;
    bool isNegative = false;

    /* Handle 0 explicitely, otherwise empty string is printed for 0 */
    if (num == 0)
    {
        str[i++] = '0';
        str[i] = '\0';
        return str;
    }

    // In standard itoa(), negative numbers are handled only with
    // base 10. Otherwise numbers are considered unsigned.
    if (num < 0 && base == 10)
    {
        isNegative = true;
        num = -num;
    }

    // Process individual digits
    while (num != 0)
    {
        int rem = num % base;
        str[i++] = (rem > 9)? (rem-10) + 'a' : rem + '0';
        num = num/base;
    }

    // If number is negative, append '-'
    if (isNegative)
        str[i++] = '-';

    str[i] = '\0'; // Append string terminator

    // Reverse the string
    reverse(str, i);

    return str;
}

__device__ int intToStr(int x, char str[], int d)
{
    int i = 0;
    while (x) {
        str[i++] = (x % 10) + '0';
        x = x / 10;
    }

    // If number of digits required is more, then
    // add 0s at the beginning
    while (i < d)
        str[i++] = '0';

    reverse(str, i);
    str[i] = '\0';
    return i;
}

// Converts a floating-point/double number to a string.
__device__ void ftoa(float n, char* res, int afterpoint)
{
    // Extract integer part
    int ipart = (int)n;

    // Extract floating part
    float fpart = n - (float)ipart;

    // convert integer part to string
    int i = intToStr(ipart, res, 0);

    // check for display option after point
    if (afterpoint != 0) {
        res[i] = '.'; // add dot

        // Get the value of fraction part upto given no.
        // of points after dot. The third parameter
        // is needed to handle cases like 233.007
        fpart = fpart * pow(10, afterpoint);

        intToStr((int)fpart, res + i + 1, afterpoint);
    }
}

__device__ int appendInt(char *data, int i) {
    char str[20];
    itoa(i,str);
    int j=0;
    while(str[j] != '\0'){
        data[j] = str[j];
        j++;
    }
    return j;
}

__device__ int appendFlt(char *data, float f) {
    char str[30];
    int precision = 5;
    ftoa(f, str, precision);
    int j=0;
    while(str[j] != '\0'){
        data[j] = str[j];
        j++;
    }
    return j;
}

__device__ int appendStr(char *data, const char *str) {
    int j=0;
    while(str[j] != '\0'){
        data[j] = str[j];
        j++;
    }
    return j;
}


