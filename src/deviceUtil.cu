#include "hip/hip_runtime.h"
//
// Created by gautam on 28/04/20.
//

#include "deviceUtil.cuh"

// __device__ whereExpr *exprArr;
// __device__ ColType *types;
// __device__ int *offset;
// __device__ int whereExprSize;

void eval2(void *row, int *offset, ColType *types, whereExpr *exprArr, void *&res, int &resType);

__device__ void printRowDevice(void *row, ColType *colTypes, int numCols) {
     int start =  0;
     char buff[100];
     int buffStart = 0;
     for (int i = 0; i < numCols; i++, start += colTypes[i].size) {
         switch (colTypes->type) {
             case TYPE_INT: {
                 int temp = *((int *) ((char *) row + start));
                 buffStart += appendInt(buff + buffStart, temp);
                 break;
             }
             case TYPE_FLOAT: {
                 float temp = *((float *) ((char *) row + start));
                 buffStart += appendFlt(buff + buffStart, temp);
                 break;
             }
             case TYPE_VARCHAR: {
                 char *temp = (char *) row + start;
                 buffStart += appendStr(buff + buffStart, temp);
                 break;
             }
             case TYPE_INVALID:
                 break;
         }
         if (i != numCols - 1) {
             buffStart += appendStr(buff + buffStart, ", ");
         }
     }
     buff[buffStart] = '\0';
     printf("%s\n", buff);
}

__device__ void eval2(void *row, int *offset, ColType *types, whereExpr *exprArr, void *&res, int &resType) {
    const int MAX_DEPTH = 50;
    int *exprStack = (int *) malloc(sizeof(int) * MAX_DEPTH);
    exprStack[0] = 0;        // Push Expr of 0
    bool *solved = (bool *) malloc(sizeof(bool) * 100);
    void **resArr = (void **) malloc(sizeof(void *) * 100);
    int *resTypeArr = (int *) malloc(sizeof(int) * 100);
    bool *freeAble = (bool *) malloc(sizeof(bool) * 100);
    for (int i = 0; i < 100; i++) {solved[i] = false; freeAble[i] = true;}
    int count = 1;
    while (count > 0) {
        // pop
        int index = exprStack[count - 1];
        whereExpr *expr = &exprArr[index];
        --count;
        // check if children are solved; if solved evaluate current node; else push children
        if(!solved[index]){
            // push this and ALL children
            exprStack[count] = index;
            ++count;
            if (expr->childLeft != -1) {
                exprStack[count] = expr->childLeft;
                ++count;
            }
            if (expr->childRight != -1) {
                exprStack[count] = expr->childRight;
                ++count;
            }
            continue;
        }
        switch (expr->type) {
            case CONSTANT_ERR:
                printf("Error");
                break;
            case CONSTANT_INT: {
                solved[index] = true;
                resTypeArr[index] = RESTYPE_INT;
                int *temp = (int *) malloc(sizeof(int));
                *temp = expr->iVal;
                resArr[index] = temp;
                break;
            }
            case CONSTANT_FLT: {
                solved[index] = true;
                resTypeArr[index] = RESTYPE_FLT;
                float *temp = (float *) malloc(sizeof(float));
                *temp = expr->fVal;
                resArr[index] = temp;
                break;
            }
            case CONSTANT_STR:{
                solved[index] = true;
                resTypeArr[index] = -myStrlen(expr->sVal) - 1;
                resArr[index] = expr->sVal;
                freeAble[index] = false;
                break;
            }
            case COL_NAME: {
                int colId = expr->iVal;
                int start = offset[colId];
                solved[index] = true;
                resArr[index] = (char *) row + start;
                freeAble[index] = false;
                switch (types[colId].type) {
                    case TYPE_INT: {
                        resTypeArr[index] = RESTYPE_INT;
                        break;
                    }
                    case TYPE_FLOAT: {
                        resTypeArr[index] = RESTYPE_FLT;
                        break;
                    }
                    case TYPE_VARCHAR: {
                        resTypeArr[index] = types[colId].size;
                        break;
                    }
                    default:
                        printf("Not implemented\n");
                }
            }
            case OPERATOR_PL: {
                if (solved[expr->childLeft] && solved[expr->childRight]) {
                    solved[index] = true;
                    int ltype = resTypeArr[expr->childLeft];
                    int rtype = resTypeArr[expr->childRight];
                    void *lres = resArr[expr->childLeft];
                    void *rres = resArr[expr->childRight];
                    resTypeArr[index] = RESTYPE_FLT;
                    if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
                        float lhs, rhs;
                        float *temp = (float *) malloc(sizeof(float));
                        lhs = *(float *) lres;
                        rhs = *(float *) rres;
                        *temp = lhs + rhs;
                        resArr[index] = temp;
                    } else if (ltype == RESTYPE_FLT) {
                        float lhs;
                        int rhs;
                        float *temp = (float *) malloc(sizeof(float));
                        lhs = *(float *) lres;
                        rhs = *(int *) rres;
                        *temp = lhs + rhs;
                        resArr[index] = temp;
                    } else if (rtype == RESTYPE_FLT) {
                        int lhs;
                        float rhs;
                        float *temp = (float *) malloc(sizeof(float));
                        lhs = *(int *) lres;
                        rhs = *(float *) rres;
                        *temp = lhs + rhs;
                        resArr[index] = temp;
                    } else {
                        resTypeArr[index] = RESTYPE_INT;
                        int *temp = (int *) malloc(sizeof(float));
                        int lhs, rhs;
                        lhs = *(int *) lres;
                        rhs = *(int *) rres;
                        *temp = lhs + rhs;
                        resArr[index] = temp;
                    }
                }
                break;
            }
            case OPERATOR_EQ: {
                if (solved[expr->childLeft] && solved[expr->childRight]) {
                    solved[index] = true;
                    int ltype = resTypeArr[expr->childLeft];
                    int rtype = resTypeArr[expr->childRight];
                    void *lres = resArr[expr->childLeft];
                    void *rres = resArr[expr->childRight];
                    resTypeArr[index] = RESTYPE_INT;
                    int *temp = (int *) malloc(sizeof(int));
                    if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
                        float lhs, rhs;
                        lhs = *(float *) lres;
                        rhs = *(float *) rres;
                        *temp = lhs == rhs;
                    } else if (ltype == RESTYPE_FLT && rtype == RESTYPE_INT) {
                        float lhs;
                        int rhs;
                        lhs = *(float *) lres;
                        rhs = *(int *) rres;
                        *temp = lhs == rhs;
                    } else if (rtype == RESTYPE_FLT && ltype == RESTYPE_INT) {
                        int lhs;
                        float rhs;
                        lhs = *(int *) lres;
                        rhs = *(float *) rres;
                        *temp = lhs == rhs;
                    } else if (ltype == RESTYPE_INT && rtype == RESTYPE_INT){
                        int lhs, rhs;
                        lhs = *(int *) lres;
                        rhs = *(int *) rres;
                        *temp = lhs == rhs;
                    } else {
                        *temp = myStrncmp((const char *) lres, (const char *) rres, min(-ltype, -rtype)) == 0;
                    }
                    resArr[index] = temp;
                }
                break;
            }
            case OPERATOR_MI: {
                if (solved[expr->childLeft] && solved[expr->childRight]) {
                    solved[index] = true;
                    int ltype = resTypeArr[expr->childLeft];
                    int rtype = resTypeArr[expr->childRight];
                    void *lres = resArr[expr->childLeft];
                    void *rres = resArr[expr->childRight];
                    resTypeArr[index] = RESTYPE_FLT;
                    if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
                        float lhs, rhs;
                        float *temp = (float *)malloc(sizeof(float));
                        lhs = *(float *) lres;
                        rhs = *(float *) rres;
                        *temp = lhs - rhs;
                        resArr[index] = temp;
                    } else if (ltype == RESTYPE_FLT) {
                        float lhs;
                        int rhs;
                        float *temp = (float *)malloc(sizeof(float));
                        lhs = *(float *) lres;
                        rhs = *(int *) rres;
                        *temp = lhs - rhs;
                        resArr[index] = temp;
                    } else if (rtype == RESTYPE_FLT) {
                        int lhs;
                        float rhs;
                        float *temp = (float *)malloc(sizeof(float));
                        lhs = *(int *) lres;
                        rhs = *(float *) rres;
                        *temp = lhs - rhs;
                        resArr[index] = temp;
                    } else {
                        resTypeArr[index] = RESTYPE_INT;
                        int *temp = (int *)malloc(sizeof(float));
                        int lhs, rhs;
                        lhs = *(int *) lres;
                        rhs = *(int *) rres;
                        *temp = lhs - rhs;
                        resArr[index] = temp;
                    }
                }
                break;
            }
            case OPERATOR_MU: {
                if (solved[expr->childLeft] && solved[expr->childRight]) {
                    solved[index] = true;
                    int ltype = resTypeArr[expr->childLeft];
                    int rtype = resTypeArr[expr->childRight];
                    void *lres = resArr[expr->childLeft];
                    void *rres = resArr[expr->childRight];
                    resTypeArr[index] = RESTYPE_FLT;
                    if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
                        float lhs, rhs;
                        float *temp = (float *)malloc(sizeof(float));
                        lhs = *(float *) lres;
                        rhs = *(float *) rres;
                        *temp = lhs * rhs;
                        resArr[index] = temp;
                    } else if (ltype == RESTYPE_FLT) {
                        float lhs;
                        int rhs;
                        float *temp = (float *)malloc(sizeof(float));
                        lhs = *(float *) lres;
                        rhs = *(int *) rres;
                        *temp = lhs * rhs;
                        resArr[index] = temp;
                    } else if (rtype == RESTYPE_FLT) {
                        int lhs;
                        float rhs;
                        float *temp = (float *)malloc(sizeof(float));
                        lhs = *(int *) lres;
                        rhs = *(float *) rres;
                        *temp = lhs * rhs;
                        resArr[index] = temp;
                    } else {
                        resTypeArr[index] = RESTYPE_INT;
                        int *temp = (int *)malloc(sizeof(float));
                        int lhs, rhs;
                        lhs = *(int *) lres;
                        rhs = *(int *) rres;
                        *temp = lhs * rhs;
                        resArr[index] = temp;
                    }
                }
                break;
            }
            case OPERATOR_MO: {
                if (solved[expr->childLeft] && solved[expr->childRight]) {
                    solved[index] = true;
                    int ltype = resTypeArr[expr->childLeft];
                    int rtype = resTypeArr[expr->childRight];
                    void *lres = resArr[expr->childLeft];
                    void *rres = resArr[expr->childRight];
                    resTypeArr[index] = RESTYPE_INT;
                    int *temp = (int *)malloc(sizeof(float));
                    int lhs, rhs;
                    lhs = *(int *) lres;
                    rhs = *(int *) rres;
                    *temp = lhs % rhs;
                    resArr[index] = temp;
                }
                break;
            }

            default:
                printf("Not yet implemented");
        }
    }
    // Result and restype are stored in resArr[0] and resArrType[0]
    res = resArr[0];
    resType = resTypeArr[0];
    free(resTypeArr);
    for (int i = 1; i < 100; i++) {
        if (solved[i] && freeAble[i]) {
            free(resArr[i]);
        }
    }
    free(resArr);
}

__device__ void eval(void *row, int *offset2, ColType types2[],
          whereExpr *exprArr2,
          void *&res, int &resType, int tid, bool flag) {
    if (tid == 0) {
        // offset = offset2;
        // types = types2;
        // exprArr = exprArr2;
        // whereExprSize = sizeof(whereExpr);
    }
    __syncthreads();
    if (tid == 1) {
        printf("Inside eval.\n");
        // for (int i = 0; i < 3; i++) {
        //     auto leaf = exprArr2[i];
        //     printf("TYPE: %d, ival: %d, fval: %f, sval: %s, left: %d, right: %d\n", leaf.type, leaf.iVal, leaf.fVal,
        //            leaf.sVal, leaf.childLeft, leaf.childRight);
        // }
    }
    if (flag) {
        // evalUtil(row, 0, res, resType);
        eval2(row, offset2, types2, exprArr2, res, resType);
    }
}



// __device__ void evalUtil(void *row, int currPos, void *&res, int &resType) {
//     printf("CURR POS: %d\n", currPos);
//     printf("WhereExpr size: %lu\n", sizeof(exprArr[0]));
//     whereExpr *expr = exprArr + currPos;
//     printf("Address: %ld\n", (long) &expr);
//     const auto leaf = *expr;
//     printf("TYPE inside Eval: %d, ival: %d, fval: %f, sval: %s, left: %d, right: %d\n", leaf.type, leaf.iVal, leaf.fVal,
//            leaf.sVal, leaf.childLeft, leaf.childRight);
//     switch (expr->type) {
//         case CONSTANT_ERR:
//             // printf("ERROR NOT SUPPORTED YET\n");
//             break;
//         case CONSTANT_INT:
//             // printf("INT_VAL\n");
//             // fflush(stdout);
//             res = malloc(sizeof(int));
//             resType = RESTYPE_INT;
//             memcpy(res, &expr->iVal, sizeof(int));
//             printf("val: %d\n", *((int *) res));
//             printf("val: %d\n", expr->iVal);
//             break;
//         case CONSTANT_FLT:
//             res = malloc(sizeof(float));
//             resType = RESTYPE_FLT;
//             memcpy(res, &expr->fVal, sizeof(float));
//             break;
//         case CONSTANT_STR: {
//             int len = myStrlen(expr->sVal);
//             res = malloc(sizeof(char) * len + 1);
//             resType = (int) (-len - 1);
//             memcpy(res, &expr->sVal, len + 1);
//             break;
//         }
//         case COL_NAME: {
//             int i = expr->iVal;
//             int start = offset[i];
//             int end = offset[i + 1];
//             switch (types[i].type) {
//                 case TYPE_INT:
//                     res = malloc(sizeof(int));
//                     resType = RESTYPE_INT;
//                     memcpy(res, (char *) row + start, sizeof(int));
//                     printf("Col val: %d\n", *(int *) res);
//                     break;
//                 case TYPE_FLOAT:
//                     res = malloc(sizeof(float));
//                     resType = RESTYPE_FLT;
//                     memcpy(res, (char *) row + start, sizeof(float));
//                     break;
//                 case TYPE_BOOL:
//                     printf("Not yet implemented\n");
//                     break;
//                 case TYPE_VARCHAR:
//                     res = malloc(end - start);
//                     resType = -(end - start + 1);
//                     memcpy(res, (char *) row + start, end - start);
//                     break;
//                 case TYPE_DATETIME:
//                     printf("Not yet implemented 2\n");
//                     break;
//                 case TYPE_INVALID:
//                     printf("INVALID TYPE!\n");
//                     break;
//             }
//             break;
//         }
//         case OPERATOR_PL: {
//             void *lres, *rres;
//             int ltype = 0, rtype = 0;
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childLeft, lres, ltype);
//             // size_t mem_free_0, mem_tot_0;
//             // hipMemGetInfo(&mem_free_0, &mem_tot_0);
//             printf("left: %d, right: %d\n", expr->childLeft, expr->childRight);
//             evalUtil(row, expr->childLeft, lres, ltype);
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childRight, rres, rtype);
//             evalUtil(row, expr->childRight, rres, rtype);;
//             resType = RESTYPE_FLT;
//             if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
//                 res = malloc(sizeof(float));
//                 float temp = 0;
//                 float lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = lhs + rhs;
//                 memcpy(res, &temp, sizeof(float));
//             } else if (ltype == RESTYPE_FLT) {
//                 res = malloc(sizeof(float));
//                 float temp = 0;
//                 float lhs;
//                 int rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = lhs + rhs;
//                 memcpy(res, &temp, sizeof(float));
//             } else if (rtype == RESTYPE_FLT) {
//                 res = malloc(sizeof(float));
//                 float temp = 0;
//                 int lhs;
//                 float rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = (lhs + rhs);
//                 memcpy(res, &temp, sizeof(float));
//             } else {
//                 res = malloc(sizeof(int));
//                 resType = RESTYPE_INT;
//                 int temp;
//                 int lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = (lhs + rhs);
//                 memcpy(res, &temp, sizeof(int));
//                 // printf("lhs: %d, rhs: %d", lhs, rhs);
//             }
//             // fflush(stdout);
//             free(lres);
//             free(rres);
//             break;
//         }
//         case OPERATOR_AND: {
//             void *lres, *rres;
//             int ltype = 0, rtype = 0;
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childLeft, lres, ltype);
//             evalUtil(row, expr->childLeft, lres, ltype);
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childRight, rres, rtype);
//             evalUtil(row, expr->childRight, rres, rtype);
//             res = malloc(sizeof(int));
//             resType = RESTYPE_INT;
//             int temp = 0;
//             if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
//                 float lhs, rhs;
//                 lhs = *(float *) lres;
//                 rhs = *(float *) rres;
//                 // memcpy(&lhs, lres, sizeof(float));
//                 // memcpy(&rhs, rres, sizeof(float));
//                 temp = lhs && rhs;
//             } else if (ltype == RESTYPE_FLT) {
//                 float lhs;
//                 int rhs;
//                 lhs = *(float *) lres;
//                 rhs = *(int *) rres;
//                 // memcpy(&lhs, lres, sizeof(float));
//                 // memcpy(&rhs, rres, sizeof(int));
//                 temp = lhs && rhs;
//             } else if (rtype == RESTYPE_FLT) {
//                 int lhs;
//                 float rhs;
//                 lhs = *(int *) lres;
//                 rhs = *(float *) rres;
//                 // memcpy(&lhs, lres, sizeof(int));
//                 // memcpy(&rhs, rres, sizeof(float));
//                 temp = lhs && rhs;
//             } else {
//                 int lhs, rhs;
//                 lhs = *(int *) lres;
//                 rhs = *(int *) rres;
//                 // memcpy(&lhs, lres, sizeof(int));
//                 // memcpy(&rhs, rres, sizeof(int));
//                 temp = lhs && rhs;
//             }
//             memcpy(res, &temp, sizeof(int));
//             // printf("INSIDE AND %d\n", temp);
//             // fflush(stdout);
//             free(lres);
//             free(rres);
//             break;
//         }
//         case OPERATOR_OR: {
//             void *lres, *rres;
//             int ltype = 0, rtype = 0;
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childLeft, lres, ltype);
//             evalUtil(row, expr->childLeft, lres, ltype);
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childRight, rres, rtype);
//             evalUtil(row, expr->childRight, rres, rtype);
//             res = malloc(sizeof(int));
//             resType = RESTYPE_INT;
//             int temp = 0;
//             if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
//                 float lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = lhs || rhs;
//             } else if (ltype == RESTYPE_FLT) {
//                 float lhs;
//                 int rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = lhs || rhs;
//             } else if (rtype == RESTYPE_FLT) {
//                 int lhs;
//                 float rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = lhs || rhs;
//             } else {
//                 int lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = lhs || rhs;
//             }
//             memcpy(res, &temp, sizeof(int));
//             // printf("INSIDE AND %d\n", temp);
//             // fflush(stdout);
//             free(lres);
//             free(rres);
//             break;
//         }
//         case OPERATOR_NOT: {
//             void *lres;
//             int ltype = 0;
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childLeft, lres, ltype);
//             evalUtil(row, expr->childLeft, lres, ltype);
//             res = malloc(sizeof(int));
//             resType = RESTYPE_INT;
//             int temp = 0;
//             if (ltype == RESTYPE_FLT) {
//                 float lhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 temp = !lhs;
//             } else {
//                 int lhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 temp = !lhs;
//                 // printf("lhs: %d, rhs: %d", lhs, rhs);
//             }
//             memcpy(res, &temp, sizeof(int));
//             // printf("INSIDE = %d\n", temp);
//             // fflush(stdout);
//             free(lres);
//             break;
//         }
//         case OPERATOR_EQ: {
//             void *lres, *rres;
//             int ltype = 0, rtype = 0;
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childLeft, lres, ltype);
//             evalUtil(row, expr->childLeft, lres, ltype);
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childRight, rres, rtype);
//             evalUtil(row, expr->childRight, rres, rtype);
//             res = malloc(sizeof(int));
//             resType = RESTYPE_INT;
//             int temp = 0;
//             if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
//                 float lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = lhs == rhs;
//             } else if (ltype == RESTYPE_FLT) {
//                 float lhs;
//                 int rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = lhs == rhs;
//             } else if (rtype == RESTYPE_FLT) {
//                 int lhs;
//                 float rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = lhs == rhs;
//             } else {
//                 int lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = lhs == rhs;
//                 // printf("lhs: %d, rhs: %d", lhs, rhs);
//             }
//             memcpy(res, &temp, sizeof(int));
//             // printf("INSIDE = %d\n", temp);
//             // fflush(stdout);
//             free(lres);
//             free(rres);
//             break;
//         }
//         case OPERATOR_NE: {
//             void *lres, *rres;
//             int ltype = 0, rtype = 0;
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childLeft, lres, ltype);
//             evalUtil(row, expr->childLeft, lres, ltype);
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childRight, rres, rtype);
//             evalUtil(row, expr->childRight, rres, rtype);
//             res = malloc(sizeof(int));
//             resType = RESTYPE_INT;
//             int temp = 0;
//             if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
//                 float lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = (lhs != rhs);
//             } else if (ltype == RESTYPE_FLT) {
//                 float lhs;
//                 int rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = (lhs != rhs);
//             } else if (rtype == RESTYPE_FLT) {
//                 int lhs;
//                 float rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = (lhs != rhs);
//             } else {
//                 int lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = (lhs != rhs);
//                 // printf("lhs: %d, rhs: %d", lhs, rhs);
//             }
//             memcpy(res, &temp, sizeof(int));
//             // printf("INSIDE = %d\n", temp);
//             // fflush(stdout);
//             free(lres);
//             free(rres);
//             break;
//         }
//         case OPERATOR_GE: {
//             void *lres, *rres;
//             int ltype = 0, rtype = 0;
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childLeft, lres, ltype);
//             evalUtil(row, expr->childLeft, lres, ltype);
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childRight, rres, rtype);
//             evalUtil(row, expr->childRight, rres, rtype);;
//             res = malloc(sizeof(int));
//             resType = RESTYPE_INT;
//             int temp = 0;
//             if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
//                 float lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = (lhs >= rhs);
//             } else if (ltype == RESTYPE_FLT) {
//                 float lhs;
//                 int rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = (lhs >= rhs);
//             } else if (rtype == RESTYPE_FLT) {
//                 int lhs;
//                 float rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = (lhs >= rhs);
//             } else {
//                 int lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = (lhs >= rhs);
//                 // printf("lhs: %d, rhs: %d", lhs, rhs);
//             }
//             memcpy(res, &temp, sizeof(int));
//             // printf("INSIDE = %d\n", temp);
//             // fflush(stdout);
//             free(lres);
//             free(rres);
//             break;
//         }
//         case OPERATOR_LE: {
//             void *lres, *rres;
//             int ltype = 0, rtype = 0;
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childLeft, lres, ltype);
//             evalUtil(row, expr->childLeft, lres, ltype);
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childRight, rres, rtype);
//             evalUtil(row, expr->childRight, rres, rtype);;
//             res = malloc(sizeof(int));
//             resType = RESTYPE_INT;
//             int temp = 0;
//             if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
//                 float lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = (lhs <= rhs);
//             } else if (ltype == RESTYPE_FLT) {
//                 float lhs;
//                 int rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = (lhs <= rhs);
//             } else if (rtype == RESTYPE_FLT) {
//                 int lhs;
//                 float rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = (lhs <= rhs);
//             } else {
//                 int lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = (lhs <= rhs);
//                 // printf("lhs: %d, rhs: %d", lhs, rhs);
//             }
//             memcpy(res, &temp, sizeof(int));
//             // printf("INSIDE = %d\n", temp);
//             // fflush(stdout);
//             free(lres);
//             free(rres);
//             break;
//         }
//         case OPERATOR_GT: {
//             void *lres, *rres;
//             int ltype = 0, rtype = 0;
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childLeft, lres, ltype);
//             evalUtil(row, expr->childLeft, lres, ltype);
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childRight, rres, rtype);
//             evalUtil(row, expr->childRight, rres, rtype);;
//             res = malloc(sizeof(int));
//             resType = RESTYPE_INT;
//             int temp = 0;
//             if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
//                 float lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = (lhs > rhs);
//             } else if (ltype == RESTYPE_FLT) {
//                 float lhs;
//                 int rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = (lhs > rhs);
//             } else if (rtype == RESTYPE_FLT) {
//                 int lhs;
//                 float rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = (lhs > rhs);
//             } else {
//                 int lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = (lhs > rhs);
//                 // printf("lhs: %d, rhs: %d", lhs, rhs);
//             }
//             memcpy(res, &temp, sizeof(int));
//             // printf("INSIDE = %d\n", temp);
//             // fflush(stdout);
//             free(lres);
//             free(rres);
//             break;
//         }
//         case OPERATOR_LT: {
//             void *lres, *rres;
//             int ltype = 0, rtype = 0;
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childLeft, lres, ltype);
//             evalUtil(row, expr->childLeft, lres, ltype);
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childRight, rres, rtype);
//             evalUtil(row, expr->childRight, rres, rtype);;
//             res = malloc(sizeof(int));
//             resType = RESTYPE_INT;
//             int temp = 0;
//             if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
//                 float lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = (lhs < rhs);
//             } else if (ltype == RESTYPE_FLT) {
//                 float lhs;
//                 int rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = (lhs < rhs);
//             } else if (rtype == RESTYPE_FLT) {
//                 int lhs;
//                 float rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = (lhs < rhs);
//             } else {
//                 int lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = (lhs < rhs);
//                 // printf("lhs: %d, rhs: %d", lhs, rhs);
//             }
//             memcpy(res, &temp, sizeof(int));
//             // printf("INSIDE = %d\n", temp);
//             // fflush(stdout);
//             free(lres);
//             free(rres);
//             break;
//         }
//         case OPERATOR_MI: {
//             void *lres, *rres;
//             int ltype = 0, rtype = 0;
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childLeft, lres, ltype);
//             evalUtil(row, expr->childLeft, lres, ltype);
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childRight, rres, rtype);
//             evalUtil(row, expr->childRight, rres, rtype);
//             res = malloc(sizeof(int));
//             resType = RESTYPE_FLT;
//             if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
//                 float temp = 0;
//                 float lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = lhs - rhs;
//                 memcpy(res, &temp, sizeof(float));
//             } else if (ltype == RESTYPE_FLT) {
//                 float temp = 0;
//                 float lhs;
//                 int rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = lhs - rhs;
//                 memcpy(res, &temp, sizeof(float));
//             } else if (rtype == RESTYPE_FLT) {
//                 float temp = 0;
//                 int lhs;
//                 float rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = (lhs - rhs);
//                 memcpy(res, &temp, sizeof(float));
//             } else {
//                 resType = RESTYPE_INT;
//                 int temp;
//                 int lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = (lhs - rhs);
//                 memcpy(res, &temp, sizeof(int));
//                 // printf("lhs: %d, rhs: %d", lhs, rhs);
//             }
//             // fflush(stdout);
//             free(lres);
//             free(rres);
//             break;
//         }
//         case OPERATOR_MU: {
//             void *lres, *rres;
//             int ltype = 0, rtype = 0;
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childLeft, lres, ltype);
//             evalUtil(row, expr->childLeft, lres, ltype);
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childRight, rres, rtype);
//             evalUtil(row, expr->childRight, rres, rtype);
//             res = malloc(sizeof(int));
//             resType = RESTYPE_FLT;
//             if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
//                 float temp = 0;
//                 float lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = lhs * rhs;
//                 memcpy(res, &temp, sizeof(float));
//             } else if (ltype == RESTYPE_FLT) {
//                 float temp = 0;
//                 float lhs;
//                 int rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = lhs * rhs;
//                 memcpy(res, &temp, sizeof(float));
//             } else if (rtype == RESTYPE_FLT) {
//                 float temp = 0;
//                 int lhs;
//                 float rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = (lhs * rhs);
//                 memcpy(res, &temp, sizeof(float));
//                 // printf("Value : !!%f!!", temp);
//             } else {
//                 resType = RESTYPE_INT;
//                 int temp;
//                 int lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(int));
//                 printf("lhs: %d, rhs: %d", lhs, rhs);
//                 temp = (lhs * rhs);
//                 memcpy(res, &temp, sizeof(int));
//             }
//             // fflush(stdout);
//             free(lres);
//             free(rres);
//             break;
//         }
//         case OPERATOR_DI: {
//             void *lres, *rres;
//             int ltype = 0, rtype = 0;
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childLeft, lres, ltype);
//             evalUtil(row, expr->childLeft, lres, ltype);
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childRight, rres, rtype);
//             evalUtil(row, expr->childRight, rres, rtype);
//             res = malloc(sizeof(int));
//             resType = RESTYPE_FLT;
//             if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
//                 float temp = 0;
//                 float lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = lhs / rhs;
//                 memcpy(res, &temp, sizeof(float));
//             } else if (ltype == RESTYPE_FLT) {
//                 float temp = 0;
//                 float lhs;
//                 int rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = lhs / rhs;
//                 memcpy(res, &temp, sizeof(float));
//             } else if (rtype == RESTYPE_FLT) {
//                 float temp = 0;
//                 int lhs;
//                 float rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = (lhs / rhs);
//                 memcpy(res, &temp, sizeof(float));
//             } else {
//                 resType = RESTYPE_INT;
//                 int temp;
//                 int lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = (lhs / rhs);
//                 memcpy(res, &temp, sizeof(int));
//                 // printf("lhs: %d, rhs: %d", lhs, rhs);
//             }
//             // fflush(stdout);
//             free(lres);
//             free(rres);
//             break;
//         }
//         case OPERATOR_MO: {
//             void *lres, *rres;
//             int ltype = 0, rtype = 0;
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childLeft, lres, ltype);
//             evalUtil(row, expr->childLeft, lres, ltype);
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childRight, rres, rtype);
//             evalUtil(row, expr->childRight, rres, rtype);
//             res = malloc(sizeof(int));
//             resType = RESTYPE_INT;
//             int temp = 0;
//             if (ltype == RESTYPE_FLT && rtype == RESTYPE_FLT) {
//                 float lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = (int) lhs % (int) rhs;
//             } else if (ltype == RESTYPE_FLT) {
//                 float lhs;
//                 int rhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = (int) lhs % rhs;
//             } else if (rtype == RESTYPE_FLT) {
//                 int lhs;
//                 float rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(float));
//                 temp = lhs % (int) rhs;
//             } else {
//                 int lhs, rhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 memcpy(&rhs, rres, sizeof(int));
//                 temp = lhs % rhs;
//                 // printf("lhs: %d, rhs: %d", lhs, rhs);
//             }
//             memcpy(res, &temp, sizeof(int));
//             // fflush(stdout);
//             free(lres);
//             free(rres);
//             break;
//         }
//         case OPERATOR_UMI: {
//             void *lres;
//             int ltype = 0;
//             //eval(row, rowSize, offset, offsetSize, types, exprArr, expr->childLeft, lres, ltype);
//             evalUtil(row, expr->childLeft, lres, ltype);
//             res = malloc(sizeof(int));
//             resType = RESTYPE_FLT;
//             if (ltype == RESTYPE_FLT) {
//                 float temp = 0;
//                 float lhs;
//                 memcpy(&lhs, lres, sizeof(float));
//                 temp = -lhs;
//                 memcpy(res, &temp, sizeof(int));
//             } else {
//                 resType = RESTYPE_INT;
//                 int temp;
//                 int lhs;
//                 memcpy(&lhs, lres, sizeof(int));
//                 temp = -lhs;
//                 memcpy(res, &temp, sizeof(int));
//                 // printf("lhs: %d, rhs: %d", lhs, rhs);
//             }
//             // fflush(stdout);
//             free(lres);
//             break;
//         }
//     }
// }

__device__ int myStrncmp(const char *str_a, const char *str_b, unsigned len) {
    int match = 0;
    unsigned i = 0;
    unsigned done = 0;
    while ((i < len) && (match == 0) && !done) {
        if ((str_a[i] == 0) || (str_b[i] == 0)) done = 1;
        else if (str_a[i] != str_b[i]) {
            match = i + 1;
            if ((int) str_a[i] - (int) str_b[i] < 0) match = 0 - (i + 1);
        }
        i++;
    }
    return match;
}

__device__ int myStrlen(const char *str) {
    int length = 0;
    while (str[length++] != 0);
    return length;
}

__device__ void reverse(char* str, int len)
{
    int i = 0, j = len - 1, temp;
    while (i < j) {
        temp = str[i];
        str[i] = str[j];
        str[j] = temp;
        i++;
        j--;
    }
}

// Implementation of itoa()
__device__ char* itoa(int num, char* str)
{
    int base = 10;
    int i = 0;
    bool isNegative = false;

    /* Handle 0 explicitely, otherwise empty string is printed for 0 */
    if (num == 0)
    {
        str[i++] = '0';
        str[i] = '\0';
        return str;
    }

    // In standard itoa(), negative numbers are handled only with
    // base 10. Otherwise numbers are considered unsigned.
    if (num < 0 && base == 10)
    {
        isNegative = true;
        num = -num;
    }

    // Process individual digits
    while (num != 0)
    {
        int rem = num % base;
        str[i++] = (rem > 9)? (rem-10) + 'a' : rem + '0';
        num = num/base;
    }

    // If number is negative, append '-'
    if (isNegative)
        str[i++] = '-';

    str[i] = '\0'; // Append string terminator

    // Reverse the string
    reverse(str, i);

    return str;
}

__device__ int intToStr(int x, char str[], int d)
{
    int i = 0;
    while (x) {
        str[i++] = (x % 10) + '0';
        x = x / 10;
    }

    // If number of digits required is more, then
    // add 0s at the beginning
    while (i < d)
        str[i++] = '0';

    reverse(str, i);
    str[i] = '\0';
    return i;
}

// Converts a floating-point/double number to a string.
__device__ void ftoa(float n, char* res, int afterpoint)
{
    // Extract integer part
    int ipart = (int)n;

    // Extract floating part
    float fpart = n - (float)ipart;

    // convert integer part to string
    int i = intToStr(ipart, res, 0);

    // check for display option after point
    if (afterpoint != 0) {
        res[i] = '.'; // add dot

        // Get the value of fraction part upto given no.
        // of points after dot. The third parameter
        // is needed to handle cases like 233.007
        fpart = fpart * pow(10, afterpoint);

        intToStr((int)fpart, res + i + 1, afterpoint);
    }
}

__device__ int appendInt(char *data, int i) {
    char str[20];
    itoa(i,str);
    int j=0;
    while(str[j] != '\0'){
        data[j] = str[j];
        j++;
    }
    return j;
}

__device__ int appendFlt(char *data, float f) {
    char str[30];
    int precision = 5;
    ftoa(f, str, precision);
    int j=0;
    while(str[j] != '\0'){
        data[j] = str[j];
        j++;
    }
    return j;
}

__device__ int appendStr(char *data, char *str) {
    int j=0;
    while(str[j] != '\0'){
        data[j] = str[j];
        j++;
    }
    return j;
}
