#include "hip/hip_runtime.h"
//
// Created by ameen on 09/05/20.
//

#include "null.cuh"

__device__ bool isNull(int *i){
    return *i == INT_MIN;
}

__device__ bool isNull(char *data){
    int i = 0;
    while (data[i] == 127) ++i;
    return data[i] == 0;
}

__device__ bool isNull(float *f){
    return isnan(*f);
}

__device__ int getNullInt(){
    return INT_MIN;
}

__device__ float getNullFlt(){
    return NAN;
}

__device__ void getNullStr(char *data, int size){
    int i=0;
    while(i < size-1){
        data[i] = 127;
        i++;
    }
    data[size-1] = 0;
}
