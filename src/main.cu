#include "hip/hip_runtime.h"
#include "CLI.cuh"
#include "Parser.cuh"
#include "deviceUtil.cuh"

//__global__ void testkernal(){
//    char str[20];
//    int size = appendInt(str, threadIdx.x);
//    int size2 = appendInt(str + size, threadIdx.x + 10);
//    str[size+size2] = '\0';
//    printf("%s", str);
//}

int main() {
    CLI interface;
    utils::loadTables();
    Parser parser;
    std::string query = interface.readLine();
    while (!query.empty()) {
        parser.parse(query);
        query = interface.readLine();
    }
    utils::writeDatabase();
    return 0;
}