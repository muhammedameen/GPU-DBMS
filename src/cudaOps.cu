#include "hip/hip_runtime.h"
//
// Created by gautam on 02/05/20.
//
#include "cudaOps.cuh"
#include "ColType.h"

__global__ void selectKernel(void *data, int rowSize, const int *offset, int offsetSize, char *cols, int *colStart, Metadata::ColType types[], whereExpr *where) {
    void *res;
    int resType = 0;
    eval(data, rowSize, offset, offsetSize, cols, colStart, types, &where[0], 0, res, resType);
    if (resType == RESTYPE_INT) {
        int *x = (int *) res;
        printf("Value of expression is : %d\n", *x);
    }
}