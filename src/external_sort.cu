//
// Created by gautam on 08/05/20.
//

#include "external_sort.cuh"

void external_sort::sort(Data &data, std::vector<std::string> &colNames) {

    std::vector<int> cols(colNames.size());
    for (int i = 0; i < colNames.size(); ++i) {
        cols[i] = data.mdata.colMap[colNames[i]];
    }

    int *cols_d;
    hipMalloc(&cols_d, sizeof(int) * cols.size());
    hipMemcpy(cols_d, &cols[0], sizeof(int) * cols.size(), hipMemcpyHostToDevice);

    void *chunk = malloc(data.mdata.rowSize * data.chunkSize);

    hipFree(cols_d);
}
