#include "hip/hip_runtime.h"
//
// Created by gautam on 07/05/20.
//

#include "sql_update.cuh"

#define NUM_THREADS 512

__global__ void
updateKernel(void *data, int rowSize, int *offset, int offsetSize, ColType *types, myExpr *exprs, int numRows,
             const int *uIds, myExpr *uExprs, int *uOffs, int numUpdates) {
    void *res;
    int resType = 1;
    int rowsPerBlock = (numRows + NUM_THREADS - 1) / NUM_THREADS;
    unsigned int start = rowsPerBlock * threadIdx.x;
    unsigned int end = rowsPerBlock * (threadIdx.x + 1);

    void *tempRow = malloc(rowSize);

    void *row;
    bool flag;
    for (unsigned int i = start; i < end; ++i) {
        if (i >= numRows) break;
        row = (char *)data + i * rowSize;
        eval(row, offset, types, exprs, res, resType);
        flag = false;
        if (resType == RESTYPE_INT) {
            flag = *(int *) res != 0;
        } else if (resType == RESTYPE_FLT) {
            flag = *(float *) res != 0;
        }
        free(res);
        if (!flag) continue;
        // update row here
        memcpy(tempRow, row, rowSize);
        for (int j = 0; j < numUpdates; ++j) {
            const int col = uIds[j];
            myExpr *uExpr = uExprs + uOffs[j];
            eval(tempRow, offset, types, uExpr, res, resType);
            switch (types[col].type) {
                case TYPE_INT:{
                    // ASSERT RESULT HAS TO BE INT
                    if (resType == RESTYPE_INT) {
                        int *x = (int *) ((char *) tempRow + offset[col]);
                        *x = *(int *) res;
                    }
                    break;
                }
                case TYPE_FLOAT: {
                    // RESULT CAN BE INT OR FLOAT
                    if (resType == RESTYPE_INT) {
                        float *x = (float *) ((char *) tempRow + offset[col]);
                        *x = *(int *) res;
                    } else if (resType == RESTYPE_FLT) {
                        float *x = (float *) ((char *) tempRow + offset[col]);
                        *x = *(float *) res;
                    }
                    break;
                }
                case TYPE_VARCHAR: {
                    // RESULT HAS TO BE VARCHAR
                    if (resType < 0 && -resType <= types[col].size) {
                        char *x = (char *) tempRow + offset[col];
                        int resEnd = appendStr(x, (char *) res);
                        x[resEnd] = 0;
                    }
                    break;
                }
                default:
                    printf("Not implemented");
                    break;
            }
        }
        memcpy(row, tempRow, rowSize);
    }
}

void sql_update::execute(std::string &query) {
    hsql::SQLParserResult *result = hsql::SQLParser::parseSQLString(query);
    std::vector<std::string> columnNames;
    std::string tableName;

    if (result->isValid()) {
        const auto *stmt = (const hsql::UpdateStatement *) result->getStatement(0);
        tableName = stmt->table->name;
        std::vector<myExpr> flattenedExpr;
        Data d(tableName);
        exprToVec(stmt->where, flattenedExpr, d.mdata.columns);

        hipSetDevice(0);
        hipDeviceReset();

        int rowSize = d.mdata.rowSize;
        void *data = malloc(d.chunkSize * rowSize);
        void *data_d;
        int numCols = d.mdata.columns.size();
        ColType *type_d;

        hipMalloc(&type_d, sizeof(ColType) * numCols);
        hipMemcpy(type_d, &d.mdata.datatypes[0], sizeof(ColType) * numCols, hipMemcpyHostToDevice);
        myExpr *where_d;
        hipMalloc(&where_d, sizeof(myExpr) * flattenedExpr.size());
        hipMemcpy(where_d, &flattenedExpr[0], sizeof(myExpr) * flattenedExpr.size(), hipMemcpyHostToDevice);
        int *offsets = (int *) malloc(sizeof(int) * (numCols + 1));
        offsets[0] = 0; //d.mdata.datatypes[0].size;
        for (int i = 1; i <= numCols; i++) {
            offsets[i] = offsets[i - 1] + d.mdata.datatypes[i - 1].size;
        }
        int *offsets_d;
        hipMalloc(&offsets_d, sizeof(int) * (numCols + 1));
        hipMemcpy(offsets_d, offsets, sizeof(int) * (numCols + 1), hipMemcpyHostToDevice);
        int numRows = d.read(data);
        hipMalloc(&data_d, d.chunkSize * rowSize);
        std::vector<std::vector<myExpr>> updateExprs(stmt->updates->size());
        std::vector<int> colIds(stmt->updates->size());
        for (int i = 0; i < stmt->updates->size(); ++i) {
            hsql::UpdateClause *clause = stmt->updates->at(i);
            colIds[i] = d.mdata.colMap[clause->column];
            exprToVec(clause->value, updateExprs[i], d.mdata.columns);
        }
        int *updateIds_d;
        hipMalloc(&updateIds_d, sizeof(int) * colIds.size());
        hipMemcpy(updateIds_d, &colIds[0], sizeof(int) * colIds.size(), hipMemcpyHostToDevice);

        myExpr *updateExprs_d;
        int total = 0;
        std::vector<int> updateOffsets(updateExprs.size());
        for (int i = 0; i < updateExprs.size(); ++i) {
            updateOffsets[i] = total;
            total += updateExprs[i].size();
        }
        hipMalloc(&updateExprs_d, sizeof(myExpr) * total);
        for (int i = 0; i < updateExprs.size(); ++i) {
            hipMemcpy(updateExprs_d + updateOffsets[i], &updateExprs[i][0], sizeof(myExpr) * updateExprs[i].size(),
                       hipMemcpyHostToDevice);
        }
        int *updateOffsets_d;
        hipMalloc(&updateOffsets_d, sizeof(int) * updateOffsets.size());
        hipMemcpy(updateOffsets_d, &updateOffsets[0], sizeof(int) * updateOffsets.size(), hipMemcpyHostToDevice);
        while (numRows > 0) {
            hipMemcpy(data_d, data, rowSize * numRows, hipMemcpyHostToDevice);
            updateKernel<<<1, NUM_THREADS>>>(data_d, rowSize, offsets_d, numCols, type_d, where_d, numRows, updateIds_d,
                                             updateExprs_d, updateOffsets_d, colIds.size());
            hipDeviceSynchronize();
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                printf("Error at %d: %s\n", __LINE__, hipGetErrorString(err));
            }
            hipMemcpy(data, data_d, rowSize * numRows, hipMemcpyDeviceToHost);
            d.write(data, numRows * d.mdata.rowSize);
            numRows = d.read(data);
        }
        // Free all the data
        free(data);
        free(offsets);
        hipFree(data_d);
        hipFree(type_d);
        hipFree(where_d);
        hipFree(offsets_d);
    } else {
        printf("QUERY is invalid\n");
    }
}